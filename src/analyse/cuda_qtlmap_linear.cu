#include "hip/hip_runtime.h"
/*
 *
 * O.Filangi : linearisation de la vraissemblance -  calcul simultané des positions sur le génome
 * -deviceemu => emulation
 * nvcc -g -arch=sm_20 --cuda analyse_whole-genome.cu
 * debugging : ddd --debugger /usr/local/cuda/bin/cuda-gdb qtlmap
 * Debug   : nvcc -g -G -gencode=arch=compute_20,code=\"sm_20,compute_20\" --cuda analyse_whole-genome.cu
 * Release : nvcc -O4 --use_fast_math  -gencode=arch=compute_20,code=\"sm_20,compute_20\" --cuda analyse_whole-genome.cu
 * 
 * 
 * TODO : XX est une matrice symetrique => utiliser que la moitie superieur de la matrice
 * 
 * pour le profiling : /usr/local/cuda/computeprof/bin/computeprof
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#include "const_gpu_impl.h"

/* Keep in memory the residual variance fitted for each Hypothesis analysis */
static DT * allVarFitted = NULL ;

void print_info_memory() {
	size_t free,total;
	hipMemGetInfo(&free, &total) ;
	printf("===============> free: %f Mb (%lu bytes)   total:%f Mb (%lu bytes) \n",double(free)/(1024*1024),free,double(total)/(1024*1024),total);
}

class  QTLMapStructDeviceDataLinear {
public:
	//DT * contingence ; /* matrice de contingence les effets statiques sont dans les premieres colonnes. */
	DT * contingence_fix ; /*                                      [NDMAX][NFIX]                         */
	DT * contingence_fix_host ;
	DT * contingence_var ; /*                                      [ND][NVAR][NPOS]                   */  
	int * constCorrIndexCol ;
	DT * Y          ; /* Vecteur de Performance Y                  [NSIM][ND]        */
	DT * CD         ; /* Vecteur des CD                            [ND]        */
	int nqtl        ; /* hypothese courante */
	int np          ; /* nombre de famille de pere */
	int nsim        ; /* nombre de simulation */
	int nposGlobal  ; /* nombre de position teste */
	int npos        ; /* nombre de position teste par block */
	int ndmax       ; /* taille du tableau Y */
	int nd          ; /* nombre de descendants a prendre en compte */
	int nLevelFix   ; /* nombre d effet non dependant de la position */
	int nLevelVar   ; /* nombre d'effet dependant de la position */
	int ntniv       ; /* nLevelFix + nLevelVar */
	double seuil_cho    ; /* seuil pour l estimation des effets */
	DT * isigsq     ; /* variance residuelle des Hypothese < NQTL           [NSIM][NP] */
	int *corIpKd    ; /* *correspondance kd -> ip (indice du pere de la progeniture kd) */

	static const int MODEL_HOMO_POLYGENIC   = 0;
	static const int MODEL_HETERO_POLYGENIC = 1;
	static const int MODEL_HOMO_ANIMAL      = 2;
public:

	QTLMapStructDeviceDataLinear();

	void init(   
			int * mode,
			int * nqtlPtr,
			DT * sigsquare_host,
			DT * xinc_d_fix,
			DT * Y_d,
			DT * CD_d,
			int *corrLevelColPtr,
			double *seuil_choPtr,
			int * ndPtr,
			int * nkdPtr,
			int * nsimPtr,
			int * npositionGlobalPtr,
			//		int * npositionBlockPtr,
			int * nLevelFixPtr,
			int * nLevelVarPtr,
			int * npPtr,
			int * sizeFamilyNp);

	virtual ~QTLMapStructDeviceDataLinear();

	void set_contingence_var(int ndmax, int nLevelVar,int nbPositionsTest, int startPosition, DT* xinc_d_var,hipStream_t * stream);

	void releaseDevice();

	size_t calculBlockPositionWorkSize(int mode) ;
} ;

class QTLMapStructDeviceWorkLinear {
public :
	DT * A_res      ; /* matrice d'incidence                                homoscedastic : [NFIX][NFIX]  , heteroscedastic:  not used ! */

	DT * XX         ; /* matrice d'incidence                                homoscedastic : [NPOS][NTNIV][NTNIV]  , heteroscedastic: [NPOS][NTNIV][NTNIV][NSIM] */
	DT * triang     ; /* matrice temporaire de la decomposition Cholesky    homoscedastic : [NPOS][NTNIV][NTNIV]  , heteroscedastic: [NPOS][NTNIV][NTNIV][NSIM] */
	DT * rhs        ; /* Vecteur RHS                                                         [NPOS][NSIM][NTNIV]  */
	hipDeviceProp_t  prop ; /* property of the GPU card used */ 

public :
	QTLMapStructDeviceWorkLinear();
	/* init de XX, triang, rhs */
	void initResolution(int mode,const QTLMapStructDeviceDataLinear & data,DT *CD,DT * sigsquare_host,int * sizeFamily);

	virtual ~QTLMapStructDeviceWorkLinear();

	void releaseDeviceResolution();


	/* recuperation du tableau vecsol stoque sur le device */
	void getAllVecsolDevice(const QTLMapStructDeviceDataLinear & data,int nbBloc,int nbThread,int *vecsol) ;

} ;


/**
 * 
 */
class QTLMapStructDeviceSolutionLinear {
public:
	DT * osigsq     ; /* Vecteur des variances residuelles                  [NPOS][NSIM][NP]     */
	DT * bestim     ; /* Vecteur des solution des effets                    [NPOS][NSIM][NTNIV ] */
	DT * lrt        ; /* rapport de vraisemblance                           [NPOS][NSIM][NP] */
public :

	QTLMapStructDeviceSolutionLinear();

	void init(const QTLMapStructDeviceDataLinear & data);

	virtual ~QTLMapStructDeviceSolutionLinear();

	void releaseDevice();

} ;


class QTLMapGenericModelCalcul {

public: 
	virtual int getType() = 0 ;
virtual void start_analysis(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) = 0 ;
virtual bool convergenceOk(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) = 0 ;
virtual void calcul_XT_X_A(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) = 0;
virtual void calcul_XT_X_B(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) = 0;
virtual void calcul_XT_X_C(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) = 0;
virtual void calcul_Cholesky_Decomposition(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) = 0;
virtual void calcul_RHS(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) = 0;
virtual void calcul_LU(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) = 0;
virtual void calcul_SIGSQ(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) = 0;
};

class QTLMapHomoscedasticModelCalcul : public QTLMapGenericModelCalcul {

public:

	QTLMapHomoscedasticModelCalcul() {};
	virtual ~QTLMapHomoscedasticModelCalcul() {};

	virtual int getType() { return QTLMapStructDeviceDataLinear::MODEL_HOMO_POLYGENIC; };

	virtual void start_analysis(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual bool convergenceOk(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution);
	virtual void calcul_XT_X_A(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_XT_X_B(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_XT_X_C(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_Cholesky_Decomposition(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_RHS(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_LU(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution);
	virtual void calcul_SIGSQ(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution);
};

class QTLMapHomoscedasticAnimalModelCalcul : public QTLMapHomoscedasticModelCalcul {
private:
	DT * M            ; /* computat from animal matrix [ND][ND] => [ I - (I + lambda.A**-1)]*/
	DT * M_h          ;
	DT * FixM         ;
public:
	QTLMapHomoscedasticAnimalModelCalcul(int nd,int nfix,DT *M_host) : QTLMapHomoscedasticModelCalcul() {
		size_t size = nd*nd*sizeof(DT);
		safecall(hipMalloc(&M,size));
		safecall(hipMemcpy(M, M_host, size, hipMemcpyHostToDevice));
		M_h = M_host;
		size = nd*nfix*sizeof(DT);
		safecall(hipMalloc(&FixM,size));
	};

	virtual ~QTLMapHomoscedasticAnimalModelCalcul() { 
		safecall(hipFree(M)); 
		safecall(hipFree(FixM)); 
	};

	virtual int getType() { return QTLMapStructDeviceDataLinear::MODEL_HOMO_ANIMAL; };

	virtual void start_analysis(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_XT_X_B(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_XT_X_C(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_RHS(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_SIGSQ(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution);
};

class QTLMapHeteroscedasticModelCalcul : public QTLMapGenericModelCalcul {
private :
	DT * lastLrtConvergenceHost ; /* save the last lrt to compute the convergence for each position/simulation */
	DT * lrtConvergenceDevice   ; /* do once an allocation to get lrt from the current analysis */
	DT * varInDevice            ; /* [NPOS][NSIM][NP]*/
public:
	/**
	 * Constructor
	 */
	QTLMapHeteroscedasticModelCalcul(int npos,int nsim,int np) { 
		size_t size=npos*nsim;
		safecall(hipHostMalloc(&lastLrtConvergenceHost,size*sizeof(DT)));
		safecall(hipMalloc(&lrtConvergenceDevice,size*sizeof(DT)));

		for (size_t i=0;i<size;i++)
			lastLrtConvergenceHost[i]=99999.0;

		size=npos*nsim*np;
		safecall(hipMalloc(&varInDevice,size*sizeof(DT)));
	} ;

	/**
	 * Destructor => delete lastlrt
	 */
	virtual ~QTLMapHeteroscedasticModelCalcul() { 
		safecall(hipHostFree(lastLrtConvergenceHost)); 
		safecall(hipFree(lrtConvergenceDevice)); 
		safecall(hipFree(varInDevice)); 
	} ;

	virtual int getType() { return QTLMapStructDeviceDataLinear::MODEL_HETERO_POLYGENIC; };

	virtual void start_analysis(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual bool convergenceOk(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution);
	virtual void calcul_XT_X_A(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_XT_X_B(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_XT_X_C(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_Cholesky_Decomposition(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_RHS(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work);
	virtual void calcul_LU(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution);
	virtual void calcul_SIGSQ(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution);
};

size_t QTLMapStructDeviceDataLinear::calculBlockPositionWorkSize(int mode) {

	/* le calcul comprend 
	 * A_Res           : NFIX^2 ou 0 (heteroscedastic)
	 * XX              : NPOS*NTNIV*NTNIV ou NPOS*NTNIV*NTNIV*NSIM (heteroscedastic)
	 * Triang          : NPOS*NTNIV*NTNIV ou NPOS*NTNIV*NTNIV*NSIM (heteroscedastic)
	 * Rhs             : NPOS*NSIM*NTNIV
	 * OSIG            : NPOS*NSIM*NP
	 * BESTIM          : NPOS*NSIM*NTNIV
	 * LRT             : NQTL*NPOSGLOBAL*NSIM
	 * CONTINGENCE_VAR : NPOS*NVAR*NPOS
	 * 
	 * 
	 * Y,CD et contingence_fix ont deja ete alloue 
	 * 
	 */

	size_t free,total,num;
	hipMemGetInfo(&free, &total) ;
	// avec la version driver 3.7 on etait obliger de diviser par 1.8 pour que ca passe.....;
	free = free / 1.05 ;


	if ( MODEL_HETERO_POLYGENIC == mode ) {
		num = ( nqtl*nposGlobal*nsim + np*nsim );
	}
	else if ( MODEL_HOMO_POLYGENIC == mode )  {
		num = ( nLevelFix*nLevelFix + nqtl*nposGlobal*nsim + np*nsim );
	} else if ( MODEL_HOMO_ANIMAL == mode )  {
		num = ( nLevelFix*(nLevelFix+ndmax) + nqtl*nposGlobal*nsim + np*nsim );
	}
	if ( free < num ) {
		cerr << "Not enough memory free:"<< free << " need:"<< num << endl ;
		exit(1);
	}
	size_t res = free - num*sizeof(DT);
	size_t denom ;
	/*  SIZE in memory for ONE POSITION :  HETEROSCEDASTIC              HOMOSCEDASTIC
	 *  contingence matrix :                                 ND x NVAR
	 *  XX                 :            NTNIV*NTNIVxNSIM                NTNIV * NTNIV
	 *  TRIANG             :                   ""                            ""
	 *  RHS                :                                 NSIM*NTNIV
	 *  OSIGSQ             :                                 NSIM * NP
	 *  
	 *  LRTCONVERGENCEDEVICE :                 NSIM                          0
	 *  VarInDevice          :               NSIM x NP                       0
	 */ 
	if ( MODEL_HETERO_POLYGENIC == mode ) {
		denom = (2*SIZEMATSYM(nsim,1,ntniv) + 2*nsim*ntniv+ 2*nsim*np + nd*nLevelVar + nsim);
	} else if ( MODEL_HOMO_POLYGENIC == mode )  {
		denom = (2*SIZEMATSYM(1,1,ntniv) + 2*nsim*ntniv+ nsim*np + nd*nLevelVar);
	} else if ( MODEL_HOMO_ANIMAL == mode )  {
		denom = (2*SIZEMATSYM(1,1,ntniv) + 2*nsim*ntniv+ nsim*np + nd*nLevelVar*2);
	}

	res = ceil(res/(denom*sizeof(DT)))+1;

	/* The CUDA methods Set_RHS_VAR have a limited block size in x : 16000 */
	if ( nLevelVar > 0) {
		int sizeCalculSet_Rhs_VAR = (16000 * MAX_BLOCKDIM_64) / nLevelVar ;
		if ( sizeCalculSet_Rhs_VAR < res ) res = sizeCalculSet_Rhs_VAR;
	}

	cout << "*******************************************************************************************************"<< endl;
	cout << "**   Maximum number of position to test in one block :"<< res << " / total number of position to test :"<< nposGlobal << " **"<< endl;
	cout << "*******************************************************************************************************"<< endl;
	return res ;

} ;

template <class T>
class Utils {
public:
	static void printFloatDeviceArray1D(int dim1,int nb1,T* array1D);
	static void printFloatDeviceArray2D(int dim1,int dim2,int nb1,int nb2,T* array2D);
	static void printFloatDeviceArray3D(int dim1,int dim2,int dim3,int nb1,int nb2,int nb3,T* array3D);
	static void printFloatDeviceArray4D(int dim1,int dim2,int dim3,int dim4,int nb1,int nb2,int nb3,int nb4,T* array4D);
	static void printFloatHostArray2D(int dim1,int dim2,int nb1,int nb2,T* array2D);
	static void printFloatHostArray3D(int dim1,int dim2,int dim3,int nb1,int nb2,int nb3,T* array3D);
	static void getArrayDeviceToHost(int nbBloc,int nbThread,int size,T * inDeviceArray, T *outHostArray,hipStream_t * stream);
};

/*******************************************************************************************************************************************/

QTLMapStructDeviceDataLinear::QTLMapStructDeviceDataLinear() {
	contingence_fix = NULL ; 
	contingence_var = NULL ;
	Y = NULL;
	CD = NULL;
	isigsq = NULL;
}


void QTLMapStructDeviceDataLinear::init(
		int * mode,
		int * nqtlPtr,
		DT * sigsquare_host,
		DT * xinc_d_fix,
		DT * Y_d,
		DT * CD_d,
		int *corrLevelColPtr,
		double *seuil_choPtr,
		int * ndPtr,
		int * nkdPtr,
		int * nsimPtr,
		int * npositionGlobalPtr,
		//		int * npositionBlockPtr,
		int * nLevelFixPtr,
		int * nLevelVarPtr,
		int * npPtr,
		int * sizeFamilyNp) {

	seuil_cho   = *seuil_choPtr;
	seuil_cho   = sqrt(seuil_cho);
	ndmax       = *ndPtr       ; /* taille reelle de la matrice de contingence : lignes */
	nd          = *nkdPtr      ; /* taille de la population pris en compte */
	nsim        = *nsimPtr     ;
	npos        = 0;
	nposGlobal  = *npositionGlobalPtr;
	nLevelFix   = *nLevelFixPtr ;
	nLevelVar   = *nLevelVarPtr;
	np          = *npPtr;
	ntniv       = nLevelFix + nLevelVar ;
	nqtl        = *nqtlPtr;

#if(_CUDA_HOST_DEBUG_)
	printf("*********************** INIT DATA *********************************\n");
	printf("mode          : %d\n",*mode);
	printf("seuil_cho  SQ : %f\n",seuil_cho);
	printf("ndmax         : %d\n",ndmax);
	printf("nd            : %d\n",nd);
	printf("nsim          : %d\n",nsim);
	printf("np            : %d\n",np);
	printf("nposGlobal    : %d\n",nposGlobal);
	printf("npos          : %d\n",npos);
	printf("nLevelFix     : %d\n",nLevelFix);
	printf("nLevelVar     : %d\n",nLevelVar);
	for (int i=0;i<np;i++) printf("sizeFamilyNp[%d] : %d\n",i,sizeFamilyNp[i]);
	for (int i=0;i<ntniv;i++) printf("corrLevelCol[%d] : %d\n",i,corrLevelColPtr[i]);
	for (int i=0;i<nd;i++) printf("Y[%d]=%f,",i,(double)Y_d[i*nsim]); printf("\n");
	for (int i=0;i<nd;i++) printf("CD[%d]=%f,",i,(double)CD_d[i]);printf("\n");
	for (int i=0;i<np;i++) printf("sigsq[%d]         : %f\n",i,(double)sigsquare_host[i*nsim]);
	printf("LAST Y[%d]        : %f\n",(nd-1)*nsim,(double)Y_d[(nd-1)*nsim]);
	printf("DEPASSEMENT 1 - LAST Y[%d]        : %f\n",(nd)*nsim,(double)Y_d[(nd)*nsim]);
#endif

	/* Y */
	size_t size = nsim*nd*sizeof(DT);
	int err = hipMalloc(&Y,size);
	if ( err ) {
		cerr << "Error allocation Y ERR="<< err << " SIZE=" << size<< endl;
		exit(1);
	}
	safecall(hipMemcpy(Y, Y_d, size, hipMemcpyHostToDevice));

	/* CD */
	size = nd*sizeof(DT);
	safecall(hipMalloc(&CD,size));
	safecall(hipMemcpy(CD, CD_d, size, hipMemcpyHostToDevice));

	/* les effets fixes */
	size = ndmax*nLevelFix*sizeof(DT) ;
	safecall(hipMalloc(&contingence_fix,size));
	safecall(hipMemcpy(contingence_fix, xinc_d_fix, size, hipMemcpyHostToDevice));

	safecall(hipHostMalloc(&contingence_fix_host,size));
	for (int i=0;i<ndmax*nLevelFix;i++) {
		contingence_fix_host[i] = xinc_d_fix[i];
	}

	if ( nLevelVar > 0 ) {
		/* isigsq */
		size = np*nsim*nqtl*sizeof(DT) ;
		safecall(hipMalloc(&isigsq,size));
		safecall(hipMemcpy(isigsq,allVarFitted, size, hipMemcpyHostToDevice));
		//		for (int isim=0;isim<nsim;isim++) {
		//			for (int ip=0;ip<np;ip++)
		//				cout << allVarFitted[isim+ip*nsim]<<" " ;
		//			cout << endl ;
		//		}
		//exit(1);
	} 

	int ip=0;
	int subtotal=sizeFamilyNp[ip];
	int corIpKd_host[nd];
	for (int kd=0;kd<nd;kd++) {
		if ( kd < subtotal ) {
			corIpKd_host[kd]=ip;
		} else {
			ip++;
			subtotal+=sizeFamilyNp[ip];
			corIpKd_host[kd]=ip;
		}					
	}

	/* corIpKd */
	size = nd*sizeof(int) ;
	safecall(hipMalloc(&corIpKd,size));
	safecall(hipMemcpy(corIpKd, corIpKd_host, size, hipMemcpyHostToDevice));
	safecall(hipGetLastError());
}


void QTLMapStructDeviceDataLinear::set_contingence_var(int ndmax, int ,int nbPositionsTest, int startPosition, DT* xinc_d_var,hipStream_t * stream) {
	if ( nLevelVar > 0 ) {
		size_t size = nd*nLevelVar*nbPositionsTest*sizeof(DT) ;
		/* 1er appel initialisation du block memoire */
		if ( contingence_var == NULL ) {
			cout << "Size of subset contingence matrix :"<< size/(1024*1024) << " Mo"<< endl ;
			/* les effets variables a chaque position */
			safecall(hipMalloc(&contingence_var,size));

		}

		/* rearrangement memoire */
		/* On peut stocker le pointeur et eviter une reallocation memoire du block temporaire pour le transfert */
		DT* n_contvar;
		safecall(hipHostMalloc(& n_contvar, size));

		for (int i=0;i<(nLevelVar*nbPositionsTest);i++) {
			for (int j=0;j<nd;j++) {
				n_contvar[j*(nLevelVar*nbPositionsTest)+i] = xinc_d_var[i*ndmax+j];
#if(_CUDA_HOST_DEBUG_)
				if ( xinc_d_var[i*ndmax+j] != xinc_d_var[i*ndmax+j] ) { // Detection des NaN
					cerr << "npos:"<< nbPositionsTest << " nlevelvar:"<< nLevelVar  << " total:"<< nLevelVar*nbPositionsTest << " nd:" << nd << " ndmax:"<< ndmax << endl ;
					cerr << "Detected NaN in xinc_d_var at position " << i << ", animal :"<< (j+1) << ":"<< n_contvar[j*(nLevelVar*nbPositionsTest)+i]<< ","<< xinc_d_var[i*ndmax+j] << endl ;
					exit(1);
				}
#endif
			}
		}

		/* */
		safecall(hipMemcpyAsync(contingence_var, n_contvar, size, hipMemcpyHostToDevice,*stream));
		safecall(hipHostFree(n_contvar));
	}
}

QTLMapStructDeviceDataLinear::~QTLMapStructDeviceDataLinear() {
}

void QTLMapStructDeviceDataLinear::releaseDevice(){
	safecall(hipFree(contingence_fix));
	if ( contingence_var != NULL ) safecall(hipFree(contingence_var));
	safecall(hipFree(Y));
	safecall(hipFree(CD));
	safecall(hipHostFree(contingence_fix_host));
	//	safecall(hipFree(sizeFam));
	if (isigsq != NULL ) safecall(hipFree(isigsq));
}


QTLMapStructDeviceWorkLinear::QTLMapStructDeviceWorkLinear() {
	XX = NULL ;
	triang = NULL ;
	rhs = NULL ;
}

void QTLMapStructDeviceWorkLinear::initResolution(int mode,const QTLMapStructDeviceDataLinear & data,DT *CD,DT * sigsquare_host,int * sizeFamily) {
	//printf("*********************** INIT WORK *********************************\n");

	/* XX */
	size_t size = (mode == QTLMapStructDeviceDataLinear::MODEL_HETERO_POLYGENIC)?SIZEMATSYM(data.nsim,data.npos,data.ntniv):SIZEMATSYM(1,data.npos,data.ntniv) ;

	if ( size > 0 ) safecall(hipMalloc(&XX,size*sizeof(DT)));

#if(_CUDA_HOST_DEBUG_)
	DT * XX_HOST ;
	safecall(hipHostMalloc(&XX_HOST,size*sizeof(DT)));
	for (size_t i=0;i<size;i++) 
		XX_HOST[i]=0.0;
	safecall(hipMemcpy(XX,XX_HOST, size*sizeof(DT), hipMemcpyHostToDevice));
	safecall(hipHostFree(XX_HOST));
#endif

	/* triang */
	if ( size > 0 ) safecall(hipMalloc(&triang,size*sizeof(DT)));

#if(_CUDA_HOST_DEBUG_)
	DT * triang_HOST ;
	safecall(hipHostMalloc(&triang_HOST,size*sizeof(DT)));
	for (size_t i=0;i<size;i++) 
		triang_HOST[i]=0.0;
	safecall(hipMemcpy(triang,triang_HOST, size*sizeof(DT), hipMemcpyHostToDevice));
	safecall(hipHostFree(triang_HOST));
#endif

	/* A_res */
	size = (mode == QTLMapStructDeviceDataLinear::MODEL_HETERO_POLYGENIC)?0:data.nLevelFix*data.nLevelFix;
	safecall(hipMalloc(&A_res,size*sizeof(DT)));

#if(_CUDA_HOST_DEBUG_)
	if ( size > 0 ) {
		DT * A_res_HOST ;
		safecall(hipHostMalloc(&A_res_HOST,size*sizeof(DT)));
		for (size_t i=0;i<size;i++) 
			A_res_HOST[i]=0.0;
		safecall(hipMemcpy(A_res,A_res_HOST, size*sizeof(DT), hipMemcpyHostToDevice));
		safecall(hipHostFree(A_res_HOST));
	}
#endif

	/* RHS */
	size = data.nsim*data.npos*data.ntniv;
	if ( size > 0 ) safecall(hipMalloc(&rhs,size*sizeof(DT)));

#if(_CUDA_HOST_DEBUG_)
	DT * rhs_HOST ;
	safecall(hipHostMalloc(&rhs_HOST,size*sizeof(DT)));

	for (int i=0;i<size;i++) 
		rhs_HOST[i]=0.0;

	safecall(hipMemcpy(rhs,rhs_HOST, size*sizeof(DT), hipMemcpyHostToDevice));
	safecall(hipHostFree(rhs_HOST));
#endif
}

QTLMapStructDeviceWorkLinear::~QTLMapStructDeviceWorkLinear() {

}

void QTLMapStructDeviceWorkLinear::releaseDeviceResolution(){
	safecall(hipFree(A_res));
	if (rhs != NULL) safecall(hipFree(rhs));
	if (XX != NULL) safecall(hipFree(XX));
	if (triang != NULL) safecall(hipFree(triang));
}


QTLMapStructDeviceSolutionLinear::QTLMapStructDeviceSolutionLinear() {
	bestim = NULL;
	osigsq = NULL;
	lrt    = NULL;
}

void QTLMapStructDeviceSolutionLinear::init(const QTLMapStructDeviceDataLinear & data) {
	/* bestim */
	size_t size = data.nsim*data.npos*data.ntniv*sizeof(DT);
	safecall(hipMalloc(&bestim,size));

	/* osigsq */
	size = data.nsim*data.npos*data.np*sizeof(DT);
	safecall(hipMalloc(&osigsq,size));

	/* lrt */
	if ( data.nqtl > 0) {
		size = data.np*data.nqtl*data.nsim*data.nposGlobal*sizeof(DT);
		safecall(hipMalloc(&lrt,size));
	} 
}

QTLMapStructDeviceSolutionLinear::~QTLMapStructDeviceSolutionLinear() {

}


void QTLMapStructDeviceSolutionLinear::releaseDevice(){
	if ( bestim != NULL ) safecall(hipFree(bestim));
	if ( osigsq != NULL ) safecall(hipFree(osigsq));
	if ( lrt != NULL ) safecall(hipFree(lrt));
}

template <class T>
void Utils<T>::printFloatDeviceArray1D(int dim1,int nb1,T* array1D) {
	size_t size = dim1*sizeof(T);

	T* array1DHost ;
	int errMH = (int) hipHostMalloc(& array1DHost, size);
	if ( errMH ) {
		printf("Error host allocation of printFloatDeviceArray1D::array1DHost ERR=%d \n",errMH);
		exit(1);
	}

	hipError_t err = hipMemcpy(array1DHost, array1D, size, hipMemcpyDeviceToHost);
	safecall(err);
	cout.precision(PRECISION_FLOTTANT);
	if ( err == 0 ) {
		cout << "****************************"<< endl;
		for (int i=0;i<nb1;i++ ) {
			cout << " " << array1DHost[i];
		}
		cout << endl <<"--"<< endl ;
		cout << "****************************"<< endl ;
	} else {
		cerr << "Error Memcpy :"<< err << endl ;
		exit(1);
	}
	safecall(hipHostFree(array1DHost));
}

template <class T>
void Utils<T>::printFloatDeviceArray2D(int dim1,int dim2,int nb1,int nb2,T* array2D) {
	size_t size = dim1*dim2*sizeof(T);

	T* array2DHost ;
	int errMH = (int) hipHostMalloc(& array2DHost, size);
	if ( errMH ) {
		printf("Error host allocation of printFloatDeviceArray2D::array2DHost ERR=%d \n",errMH);
		exit(1);
	}

	hipError_t err = hipMemcpy(array2DHost, array2D, size, hipMemcpyDeviceToHost);
	safecall(err);
	cout.precision(PRECISION_FLOTTANT);
	if ( err == 0 ) {
		cout << "****************************"<< endl ;
		for (int i=0;i<nb1;i++ ) {
			for (int j=0;j<nb2;j++ ) {
				cout << " " << array2DHost[dim1*j+i];
			}
			cout << endl ;
		}
		cout << endl <<"--"<< endl ;
		cout << "****************************"<< endl ;
	} else {
		cerr << "Error Memcpy :"<< err << endl ;
		exit(1);
	}
	safecall(hipHostFree(array2DHost));
}

template <class T>
void Utils<T>::printFloatDeviceArray3D(int dim1,int dim2,int dim3,int nb1,int nb2,int nb3,T* array3D) {
	size_t size = dim1*dim2*dim3*sizeof(T);
	T* array3DHost ;
	int errMH = (int) hipHostMalloc(& array3DHost, size);
	if ( errMH ) {
		printf("Error host allocation of printFloatDeviceArray3D::array3DHost ERR=%d \n",errMH);
		exit(1);
	}

	hipError_t err = hipMemcpy(array3DHost, array3D, size, hipMemcpyDeviceToHost);
	safecall(err);
	cout.precision(PRECISION_FLOTTANT);
	if ( err == 0 ) {
		cout << "****************************"<< endl ;
		for (int i=0;i<nb1;i++ ) {
			for (int j=0;j<nb2;j++ ) {
				for (int k=0;k<nb3 ;k++ ) {
					cout << " " << array3DHost[dim1*dim2*k+dim1*j+i];
				}
				cout << endl ;
			}
			cout << endl <<"--"<< endl ;
		}
		cout << "****************************"<< endl ;
	} else {
		cerr << "Error Memcpy :"<< err << endl ;
		exit(1);
	}
	safecall(hipHostFree(array3DHost));
}

template <class T>
void Utils<T>::printFloatDeviceArray4D(int dim1,int dim2,int dim3,int dim4,int nb1,int nb2,int nb3,int nb4,T* array4D) {
	size_t size = dim1*dim2*dim3*dim4*sizeof(T);
	T* array4DHost ;
	int errMH = (int) hipHostMalloc(& array4DHost, size);
	if ( errMH ) {
		printf("Error host allocation of printFloatDeviceArray4D::array4DHost ERR=%d \n",errMH);
		exit(1);
	}
	cout.precision(PRECISION_FLOTTANT);
	hipError_t err = hipMemcpy(array4DHost, array4D, size, hipMemcpyDeviceToHost);
	safecall(err);
	if ( err == 0 ) {
		cout << "****************************"<< endl ;
		for (int i=0;i<nb1;i++ ) {
			for (int j=0;j<nb2;j++ ) {
				for (int k=0;k<nb3 ;k++ ) {
					for (int l=0;l<nb4 ;l++ ) {
						cout << " " << array4DHost[l*dim1*dim2*dim3+dim1*dim2*k+dim1*j+i];
					}
					cout << endl ;
				}
				cout << endl << endl;
			}
			cout << endl << " --*****************************************************************************-- "<<endl;
		}
		cout << "****************************"<< endl ;
	} else {
		cerr << "Error Memcpy :"<< err << endl ;
		exit(1);
	}
	safecall(hipHostFree(array4DHost));
}


template <class T>
void Utils<T>::printFloatHostArray2D(int dim1,int dim2,int nb1,int nb2,T* array2D) {

}

template <class T>
void Utils<T>::printFloatHostArray3D(int dim1,int dim2,int dim3,int nb1,int nb2,int nb3,T* array3D) {

}

template <class T>
void Utils<T>::getArrayDeviceToHost(int nbBloc,int nbThread,int size,T * inDeviceArray, T *outHostArray,hipStream_t * stream) {
	hipError_t err ;
	if ( stream != NULL ) {
		err = hipMemcpyAsync(outHostArray,inDeviceArray, size*sizeof(T), hipMemcpyDeviceToHost,*stream);
	} else {
		err = hipMemcpy(outHostArray,inDeviceArray, size*sizeof(T), hipMemcpyDeviceToHost);
	}
	safecall(err);
	if ( err ) {
		printf("Error QTLMapStructDeviceWorkLinear::getArrayDeviceToHost = %d \n",err);
		exit(1);
	}
}



void printMatSymXXTriang(int mode,QTLMapStructDeviceDataLinear data,DT * w) {

	int sSim=(mode == QTLMapStructDeviceDataLinear::MODEL_HETERO_POLYGENIC)?data.nsim:1;
	sSim=1;
#if(!OPTIMIZE_MEMORY_MATSYM)
	size_t size = (mode == QTLMapStructDeviceDataLinear::MODEL_HETERO_POLYGENIC)?data.ntniv*data.ntniv*data.npos*data.nsim:data.ntniv*data.ntniv*data.npos;
#else
	size_t size = (mode == QTLMapStructDeviceDataLinear::MODEL_HETERO_POLYGENIC)?(CEIL(data.ntniv*(data.ntniv+1))/2)*data.npos*data.nsim:((data.ntniv*(data.ntniv+1))/2)*data.npos;
#endif	
	DT* array ;
	safecall(hipHostMalloc(& array, size));
	safecall(hipMemcpy(array, w, size, hipMemcpyDeviceToHost));

	cout.precision(PRECISION_FLOTTANT);
	for (int isim=0;isim<sSim;isim++) {
		cout << endl << "************ ISIM:"<< isim << endl;
		for (int ipos=0;ipos<1;ipos++) {
			cout << " **> POS:"<< ipos << endl << "****"<< endl ;
			for (int iniv=0;iniv<data.ntniv;iniv++) {
				for (int jniv=0;jniv<data.ntniv;jniv++) {
					cout << array[GETACC(data,isim,ipos,iniv,jniv)]<<" ";
				}
				cout << endl ;
			}
		}
	}
	safecall(hipHostFree(array));
}


/**************************************************** MODEL METHODS ******************************************************************************************/

void QTLMapHomoscedasticModelCalcul::start_analysis(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work){
	/* calcul de XX' pour la partie des effets fixe a la position sur le host */
	size_t size = data.nLevelFix*data.nLevelFix;
	DT A_res_host[size];

	for (size_t i=0;i<data.nLevelFix;i++) {
		for (size_t j=i;j<data.nLevelFix;j++) {
			A_res_host[j*data.nLevelFix+i] = 0.0;
			for (size_t kd=0;kd<data.nd;kd++) {
				A_res_host[j*data.nLevelFix+i] += data.contingence_fix_host[i*data.ndmax+kd]*data.contingence_fix_host[j*data.ndmax+kd];
			}
		}
	}

	safecall(hipMemcpy(work.A_res, A_res_host, size*sizeof(DT), hipMemcpyHostToDevice));
}

void QTLMapHomoscedasticAnimalModelCalcul::start_analysis(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work){

	/* calcul de X.M.X' pour la partie des effets fixe a la position sur le host */
	size_t size = data.nLevelFix*data.nLevelFix;
	DT A_res_host[size];

	for (size_t i=0;i<data.nLevelFix;i++) {
		for (size_t j=i;j<data.nLevelFix;j++) {
			A_res_host[j*data.nLevelFix+i] = 0.0;
			for (size_t kd=0;kd<data.nd;kd++) {
				DT v2=0;
				// Calcul de Xt . M
				for (size_t kd2=0;kd2<data.nd;kd2++ ) {
					v2 += data.contingence_fix_host[i*data.ndmax+kd2]*M_h[kd2*data.nd+kd];
				}

				A_res_host[j*data.nLevelFix+i] += v2*data.contingence_fix_host[j*data.ndmax+kd];
			}
		}
	}
	safecall(hipMemcpy(work.A_res, A_res_host, size*sizeof(DT), hipMemcpyHostToDevice));	

	DT FixM_h[data.nLevelFix*data.nd];

	/* Calcul de l'entite Colonne Fix * M =>   FixM [NFIX][ND] */
	for (size_t i=0;i<data.nLevelFix;i++) {
		for (size_t kd=0;kd<data.nd;kd++) {
			FixM_h[kd*data.nLevelFix+i] =0.0;
			// Calcul de Xt . M
			for (size_t kd2=0;kd2<data.nd;kd2++ ) {
				FixM_h[kd*data.nLevelFix+i] += data.contingence_fix_host[i*data.ndmax+kd2]*M_h[kd2*data.nd+kd];
			}
			//cout << kd << ","<< i << ":"<< FixM_h[kd*data.nLevelFix+i] << endl ;
		}
	}
	safecall(hipMemcpy(FixM, FixM_h, (data.nd*data.nLevelFix)*sizeof(DT), hipMemcpyHostToDevice));	
}



__global__ void initialize_start_variance_H0(QTLMapStructDeviceDataLinear data, DT * varInDevice) {
	int isim = blockIdx.x * blockDim.x + threadIdx.x ;

	if ( isim < data.nsim ) {

		for (size_t ip=0;ip<data.np;ip++) {
			DT somyp = 0;
			int effp = 0;
			DT var   = 0;	


			for (size_t kd=0;kd<data.nd;kd++ ) {
				if ( ip == data.corIpKd[kd] )	{
					somyp = somyp + data.Y[isim+kd*data.nsim]*data.CD[kd];
					effp++;
				}
			}

			/* mean */
			DT mu = somyp / DT(effp);

			/* variance */
			for (size_t kd=0;kd<data.nd;kd++ ) {
				if ( ip == data.corIpKd[kd] ) {
					DT v = data.Y[isim+kd*data.nsim] - mu ;
					var = var + v*v;
				}
			}

			var = var / (DT(effp) - 1);
			varInDevice[isim+ip*data.nsim] = var;
		}
	}

}

/**
 * Initialize variance to start the model (classical variance estimator)
 * Calcul de XX pour les fixe a la position
 */
__global__ void initialize_start_variance(QTLMapStructDeviceDataLinear data,DT * varInDevice) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ;
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;	

	if ( ipos < data.npos && isim < data.nsim ) {
		for (int ip=0;ip<data.np;ip++) {
			int k = (data.nqtl-1)*data.nsim*data.np;
			varInDevice[ipos+isim*data.npos+ip*data.npos*data.nsim] = data.isigsq[isim+ip*data.nsim+k] ;
		}
	}
}

void QTLMapHeteroscedasticModelCalcul::start_analysis(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work){
	/* compute start variance for analysis under H0 */


	if ( data.nqtl == 0 ) {
		dim3 dimBlock(BLOCKDIMX);
		dim3 dimGrid(ceil(data.nsim / dimBlock.x )+1);
		initialize_start_variance_H0<<<dimGrid,dimBlock,0,stream>>>(data,varInDevice);

	} else {


		//		int nbBlockY=1;
		//		while ( work.prop.maxGridSize[1] < ceil(data.nsim / nbBlockY )+1 ) {
		//			nbBlockY++;
		//		}
		//
		//		dim3 dimBlock(BLOCKDIMX,nbBlockY);
		dim3 dimBlock(32,16);
		if ( work.prop.maxGridSize[0] < ceil(data.npos / dimBlock.x )+1 ) {
			cerr << "QTLMap can not support this number of position to tested..." << endl ;
			exit(1);
		}

		dim3 dimGrid(ceil(data.npos / dimBlock.x )+1,ceil(data.nsim / dimBlock.y )+1);
		initialize_start_variance<<<dimGrid,dimBlock,0,stream>>>(data,varInDevice);

	}

	hipDeviceSynchronize();

#if(_CUDA_HOST_DEBUG_)
	hipDeviceSynchronize();
	printf("PARTIEL IVAR0\n------------POS=1,SIM=1,..,nsim => NP----------------------------------------------\n");
	Utils<DT>::printFloatDeviceArray3D(data.npos,data.nsim,data.np,data.npos,1,data.np,varInDevice);
#endif
}

void QTLMapHomoscedasticModelCalcul::calcul_XT_X_A(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {
	/* nothing to do */
}




__global__ void initialize_A_res(QTLMapStructDeviceDataLinear data,DT * XX,DT * varInDevice) {


	int ipos = blockIdx.x * blockDim.x + threadIdx.x ;
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;	

	if ( ipos < data.npos && isim < data.nsim ) {

		/* calcul de XX' des effets fixes a la position */
		for (int i=0;i<data.nLevelFix;i++) {
			for (int j=i;j<data.nLevelFix;j++) {
				DT v = 0;
				for (int kd=0;kd<data.nd;kd++) {
					v += data.contingence_fix[i*data.ndmax+kd]*data.contingence_fix[j*data.ndmax+kd]*(data.CD[kd]/varInDevice[ipos+isim*data.npos+data.corIpKd[kd]*data.npos*data.nsim]);
				}
				XX[GETACC(data,isim,ipos,data.constCorrIndexCol[j],data.constCorrIndexCol[i])] = v;
				//work.XX[GETACC(data,isim,ipos,data.constCorrIndexCol[i],data.constCorrIndexCol[j])] = v;
			}
		}
	}
}

void QTLMapHeteroscedasticModelCalcul::calcul_XT_X_A(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {

	/* sur les M2050 la limitation est de 65535, factuer limitant pour les simulations 
	 * si l utilisateur demande plus, on cree des blocks pour les simuls...*/
	//	int nbBlockY=1;
	//	while ( work.prop.maxGridSize[1] < ceil(data.nsim / nbBlockY )+1 ) {
	//		nbBlockY++;
	//	}
	//dim3 dimBlock(BLOCKDIMX,nbBlockY);
	dim3 dimBlock(32,16);

	if ( work.prop.maxGridSize[0] < ceil(data.npos / dimBlock.x )+1 ) {
		cerr << "QTLMap can not support this number of position to tested..." << endl ;
		exit(1);
	}

	dim3 dimGrid(ceil(data.npos / dimBlock.x )+1,ceil(data.nsim / dimBlock.y )+1);
	initialize_A_res<<<dimGrid,dimBlock,0,stream>>>(data,work.XX,varInDevice);
}

bool QTLMapHomoscedasticModelCalcul::convergenceOk(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) {
	return true;
}


__global__ void calcul_lrt_and_initialize_start_variance(QTLMapStructDeviceDataLinear data,QTLMapStructDeviceSolutionLinear solution,DT * varInDevice,DT * lrtConv) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ;
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;

	if ( ipos < data.npos && isim < data.nsim ) {
		DT v = 0.0; 
		for (int ip=0;ip<data.np;ip++) {
			v += constSizeFam[ip] * (log(solution.osigsq[ipos+isim*data.npos+ip*data.npos*data.nsim]) - log(varInDevice[ipos+isim*data.npos+ip*data.npos*data.nsim]));
			varInDevice[ipos+isim*data.npos+ip*data.npos*data.nsim] = solution.osigsq[ipos+isim*data.npos+ip*data.npos*data.nsim];
		}
		lrtConv[ipos+isim*data.npos] = v;
	}
}

bool QTLMapHeteroscedasticModelCalcul::convergenceOk(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) {

	//	int nbBlockY=1;
	//	while ( work.prop.maxGridSize[1] < ceil(data.nsim / nbBlockY )+1 ) {
	//		nbBlockY++;
	//	}

	//dim3 dimBlock(BLOCKDIMX,nbBlockY);
	dim3 dimBlock(32,16);

	if ( work.prop.maxGridSize[0] < ceil(data.npos / dimBlock.x )+1 ) {
		cerr << "QTLMap can not support this number of position to tested..." << endl ;
		exit(1);
	}

	dim3 dimGrid(ceil(data.npos / dimBlock.x )+1,ceil(data.nsim / dimBlock.y )+1);
	calcul_lrt_and_initialize_start_variance<<<dimGrid,dimBlock,0,stream>>>(data,solution,varInDevice,lrtConvergenceDevice);
	hipDeviceSynchronize();

	//
	size_t size=data.npos*data.nsim;
	DT  *lrt;
	//safecall(hipHostMalloc(&lrt,size*sizeof(DT)));
	lrt = new DT[size];

	safecall(hipMemcpyAsync(lrt,lrtConvergenceDevice, size*sizeof(DT), hipMemcpyDeviceToHost,stream));
	int countOk=0;

	for (int ipos=0;ipos<data.npos;ipos++) {
		for (int isim=0;isim<data.nsim;isim++) {
			int i = ipos + isim*data.npos ;
			if ( fabs(lastLrtConvergenceHost[i] - lrt[i]) < 0.5 )  countOk++;
			//			else
			//				cout << "ipos:"<< ipos << " isim:"<< isim << " "<< lrt[i] << " "<< lastLrtConvergenceHost[i] << " diff:" << fabs(lastLrtConvergenceHost[i] - lrt[i])<< endl ;
			lastLrtConvergenceHost[i] = lrt[i] ;
		}
	}

	delete [] lrt;
	//safecall(hipHostFree(lrt));

	if ( countOk ==  data.nsim*data.npos ) {
		cout << "ok" << endl ;
		return true;
	}

	cout << "convergence :" << countOk << "/"<< data.nsim*data.npos << endl ;

	return false;
}

/**********************************************************************************************************************************************
 * NFIX  : nombre de niveau fixe a la position (moyenne general, effet polygenique, effets de nuisances)
 * NVAR  : nombre de niveaux varaible a la position (effet qtl ou haplotype)
 * NTNIV : NFIX + NVAR 
 * NPOS  : nombre de position teste 
 * X     : matrice de contingence (animal par ligne / niveau exprime en colonne)
 * XT    : transpose de X
 * XT.X => Calcul independant de block A ( NFIX * NFIX operation ), B ( NFIX * NVAR * NPOS) , C (NVAR * NVAR/2 * NPOS)
 * 
 * Exemple : NFIX = 4, NVAR = 3   
 * 
 *              LEV FIX     POS1        POS2        POS3          POS4
 *           ( 1 1 0 0  0.5 0  0    0.4  0  0    0.3  0  0    0.2  0  0  )      NPOS = 4
 * TOTAL_X = ( 1 1 0 0  0.5 0  0    0.4  0  0    0.3  0  0    0.2  0  0  )
 *           ( 1 0 1 0  0  0.5 0     0  0.4  0    0  0.4  0    0  0.4  0 )    
 *           ( 1 0 1 0  0  0.5 0     0  0.4  0    0  0.4  0    0  0.4  0 )
 *           ( 1 0 0 1  0   0 0.5    0  0  0.4    0  0   0.4   0  0  0.4 )
 *           ( 1 0 0 1  0   0 0.5    0  0  0.4    0  0   0.4   0  0  0.4 )
 * 
 * pour POS = 1
 * 
 *     ( 1 1 0 0 0.5 0  0  )
 * X = ( 1 1 0 0 0.5 0  0  )
 *     ( 1 0 1 0  0 0.5 0  )     ND=6
 *     ( 1 0 1 0  0 0.5 0  )
 *     ( 1 0 0 1  0  0 0.5 )
 *     ( 1 0 0 1  0  0 0.5 )
 * 
 * 
 * XT.X = ( 1.1 1.2 1.3 1.4 1.5 1.6 1.7 )
 *        (     2.2 2.3 2.4 2.5 2.6 2.7 )
 *        (         3.3 3.4 3.5 3.6 3.7 )            ( 1.1 1.2 1.3 1.4 )         ( 1.5 1.6 1.7 )        ( 5.5 5.6 5.7 ) 
 *        (             4.4 4.5 4.6 4.7 )   avec A = (     2.2 2.3 2.4 )    B  = ( 2.5 2.6 2.7 )    C = (     6.6 6.7 )
 *        (                 5.5 5.6 5.7 )            (         3.3 3.4 )         ( 3.5 3.6 3.7 )        (         7.7 )
 *        (                     6.6 6.7 )            (             4.4 )         ( 4.5 4.6 4.7 )        
 *        (                         7.7 )
 * 
 * 
 *  - Calcul de A sur le HOST
 *  - Calcul de B sur le DEVICE
 *  - Calcul de C sur le DEVICE
 *  
 *  
 *  Calcul B :
 *  BLOCK à 2 dimensions : ivar => numero de la colonne dans la liste des niveaux variables
 *                         ifix => numero de la colonne dans la liste des niveaux fixes
 *                         
 *  
 *  Ordonnancement des resultats :
 *  [ifix] => |  1.5 pos1  | 1.6 pos1 |  1.7 pos1 |  1.5 pos2 |   | 1.6 pos2 |  1.7 pos2 ....
 *  
 */


__global__ void XT_X_B_homoscedastic(QTLMapStructDeviceDataLinear data,DT * XX) {

	extern __shared__ DT FIXVALUE[];

	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;

	if ( blockIdx.y < constNbLevelFix ) {
		int i_shared = threadIdx.x;
		while(i_shared < data.nd)
		{
			FIXVALUE[i_shared] = data.contingence_fix[blockIdx.y*data.ndmax+i_shared];
			i_shared += blockDim.x;
		}
		__syncthreads();
	}


	if ( ivar < constNbLevelVar*data.npos ) {
		DT v = 0;
		for (int kd=0;kd<data.nd;kd++ ) {
			v += FIXVALUE[kd]*data.contingence_var[kd*constNbLevelVar*data.npos+ivar];
		}

		/* niveau à une position */
		int iniv = ivar % constNbLevelVar ;
		/* position teste */
		int ipos = ivar / constNbLevelVar ;

		XX[GETACC(data,0,ipos,data.constCorrIndexCol[constNbLevelFix + iniv],data.constCorrIndexCol[blockIdx.y])] = v;
	}
}
void QTLMapHomoscedasticModelCalcul::calcul_XT_X_B(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {

	dim3 dimBlock(BLOCKDIMX);
	dim3 dimGrid (ceil(data.npos*data.nLevelVar / dimBlock.x) + 1 , (data.nLevelFix / dimBlock.y) + 1);
	XT_X_B_homoscedastic<<<dimGrid,dimBlock,data.nd*sizeof(DT),stream>>>(data,work.XX);
	safecall(hipGetLastError());
}

//******************************************************
__global__ void XT_X_B_homoscedastic_animal(QTLMapStructDeviceDataLinear data,DT * FixM,DT* M ,DT * XX) {

	//ivar : indice du niveau pour toute les positions (0<ivar<ntniv_var*npos)
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;
	// inivfix : iniv fix dans contingence fix
	int inivfix = blockIdx.y;
	if ( ivar < constNbLevelVar*data.npos ) {
		/* indice du niveau dans une matrice de contingence independant de la position (0<iniv<ntniv) */
		int iniv = ivar % constNbLevelVar ;
		/* position teste */
		int ipos = ivar / constNbLevelVar ;

		DT v = 0;
		for (int kd=0;kd<data.nd;kd++ ) {
			v += FixM[kd*constNbLevelFix+inivfix]*data.contingence_var[kd*constNbLevelVar*data.npos+ivar];
		}
		XX[GETACC(data,0,ipos,data.constCorrIndexCol[constNbLevelFix + iniv],data.constCorrIndexCol[inivfix])] = v;
	}
}

void QTLMapHomoscedasticAnimalModelCalcul::calcul_XT_X_B(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {

	dim3 dimBlock(BLOCKDIMX);
	dim3 dimGrid (ceil(data.npos*data.nLevelVar / dimBlock.x) + 1 , (data.nLevelFix / dimBlock.y) + 1);


	XT_X_B_homoscedastic_animal<<<dimGrid,dimBlock,0,stream>>>(data,FixM,M,work.XX);
	safecall(hipGetLastError());
}



__global__ void XT_X_B_heteroscedastic(DT * varInDevice,QTLMapStructDeviceDataLinear data,DT * XX) {

	extern __shared__ DT FIXVALUE[];

	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;
	int inivfix = blockIdx.z ;

	if ( inivfix < constNbLevelFix ) {
		int i_shared = threadIdx.x;
		while(i_shared < data.nd)
		{
			FIXVALUE[i_shared] = data.contingence_fix[inivfix*data.ndmax+i_shared];
			i_shared += blockDim.x;
		}
		__syncthreads();
	}


	if ( ivar < constNbLevelVar*data.npos && isim < data.nsim) {
		/* niveau à une position */
		int iniv = ivar % constNbLevelVar ;
		/* position teste */
		int ipos = ivar / constNbLevelVar ;

		DT v = 0.0;
		for (int kd=0;kd<data.nd;kd++ ) {
			v += FIXVALUE[kd]*data.contingence_var[kd*constNbLevelVar*data.npos+ivar]*(data.CD[kd] / varInDevice[ipos+isim*data.npos+data.corIpKd[kd]*data.nsim*data.npos]) ; 
		}

		XX[GETACC(data,isim,ipos,data.constCorrIndexCol[constNbLevelFix + iniv],data.constCorrIndexCol[inivfix])] = v;
		//work.XX[GETACC(data,isim,ipos,data.constCorrIndexCol[inivfix],data.constCorrIndexCol[constNbLevelFix + iniv])] = v;
	}
}

void QTLMapHeteroscedasticModelCalcul::calcul_XT_X_B(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {

	int nbBlockY=1;
	while ( work.prop.maxGridSize[1] < ceil(data.nsim / nbBlockY )+1 ) {
		nbBlockY++;
	}

	dim3 dimBlock(BLOCKDIMX,nbBlockY);

	if ( work.prop.maxGridSize[0] < ceil(data.npos*data.nLevelVar / dimBlock.x) + 1 ) {
		cerr << "QTLMap can not support this number of position ["<< data.npos << "] to tested with the number of lever ["<< data.nLevelVar <<"]" << endl ;
		exit(1);
	}

	dim3 dimGrid (ceil(data.npos*data.nLevelVar / dimBlock.x) + 1 , ceil(data.nsim / dimBlock.y )+1, data.nLevelFix);
	XT_X_B_heteroscedastic<<<dimGrid,dimBlock,data.nd*sizeof(DT),stream>>>(varInDevice,data,work.XX);
}

/* calcul d'une valeur de la diagonale et des valeurs a droite de cette diagonale de XT.X   = colonne et ligne de niveau variable =  */
/*
 * 
 *     ( 5.5 5.6 5.7 )
 * C = (     6.6 6.7 )
 *     (         7.7 )
 *     
 *     
 *     stockage du resultat dans C_res sous la forme
 *     
 *            POS 1              POS2                   POS N
 *     | 5.5 | 6.6 | 7.7 | 5.5 | 6.6 | 7.7 | ... | 5.5 | 6.6 | 7.7 |
 *     puis
 *            POS 1     POS 2            POS N
 *     | 5.6 | 6.7 | X | 5.6 | 6.7 | X | ... | 5.6 | 6.7 | X |
 *     puis
 *       POS1              POS2               POS N
 *     [ 5.7 | X | X | 5.7 | X | X |.... | 5.7 | X | X |  
 * 
 */
__global__ void XT_X_C_homoscedastic(QTLMapStructDeviceDataLinear data,DT * XX) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;

	/* ivar : la diagonale de XT . X */

	if ( ivar < constNbLevelVar*data.npos ) {	           
		/* iniv position entre 0 et ntlevVar */ 
		int nfix = constNbLevelFix ;
		int nvar = constNbLevelVar ;

		/* niveau à une position */
		int iniv = ivar % nvar ;
		/* position teste */
		int ipos = ivar / nvar ;

		/* initialisation */
		//DT viniv[MAXND_WORK] ;
		DT v=0.0,v2=0.0;

		for (int kd=0;kd<data.nd;kd++ ) {
			//viniv[kd] = data.contingence_var[kd*nvar*data.npos+ivar];
			//v += viniv[kd]*viniv[kd];	
			v2=data.contingence_var[kd*nvar*data.npos+ivar];
			v += v2*v2;
		}		

		/* calcul de la diagonal */

		/* les diagonales sont stockees en premieres...*/ 

		XX[GETACC(data,0,ipos,data.constCorrIndexCol[nfix + iniv],data.constCorrIndexCol[nfix + iniv])] = v;

		for (int jniv=1;jniv<(nvar-iniv);jniv++) {
			DT v = 0.0;	
			for (int kd=0;kd<data.nd;kd++ ) {
				//v +=  viniv[kd]*data.contingence_var[kd*nvar*data.npos+(ivar+jniv)];
				v += data.contingence_var[kd*nvar*data.npos+ivar]*data.contingence_var[kd*nvar*data.npos+(ivar+jniv)];
			}

			XX[GETACC(data,0,ipos,data.constCorrIndexCol[nfix + jniv + iniv],data.constCorrIndexCol[nfix + iniv])] = v;
			//work.XX[GETACC(data,0,ipos,data.constCorrIndexCol[nfix + iniv],data.constCorrIndexCol[nfix + jniv + iniv])] = v;
		}
	}
}


void QTLMapHomoscedasticModelCalcul::calcul_XT_X_C(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {
	dim3 dimBlock(BLOCKDIMX);
	dim3 dimGrid2 (ceil(data.npos*data.nLevelVar / dimBlock.x) + 1 , 1);
	XT_X_C_homoscedastic<<<dimGrid2,dimBlock,0,stream>>>(data,work.XX);
}


__global__ void XT_X_C_homoscedastic_animal_init(QTLMapStructDeviceDataLinear data,DT * XX) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;
	/* ivar : la diagonale de XT . X */
	if ( ivar < constNbLevelVar*data.npos ) {	           
		/* iniv position entre 0 et ntlevVar */ 
		int nfix = constNbLevelFix ;

		/* niveau à une position */
		int iniv = ivar % constNbLevelVar ;
		/* position teste */
		int ipos = ivar / constNbLevelVar ;

		XX[GETACC(data,0,ipos,data.constCorrIndexCol[nfix + iniv],data.constCorrIndexCol[nfix + iniv])] = 0.0;

		for (int jniv=1;jniv<(constNbLevelVar-iniv);jniv++) {
			XX[GETACC(data,0,ipos,data.constCorrIndexCol[nfix + jniv + iniv],data.constCorrIndexCol[nfix + iniv])] = 0.0;
		}
	}


}

__global__ void XT_X_C_homoscedastic_animal(QTLMapStructDeviceDataLinear data,int ikd,DT * M,DT * XX) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;

	/* ivar : la diagonale de XT . X */
	if ( ivar < constNbLevelVar*data.npos && ikd < data.nd ) {	           
		/* iniv position entre 0 et ntlevVar */ 
		int nfix = constNbLevelFix ;
		int nvar = constNbLevelVar ;

		/* niveau à une position */
		int iniv = ivar % nvar ;
		/* position teste */
		int ipos = ivar / nvar ;

		DT v=0;/* Calcul de XT.M */
		for (int kd=0;kd<data.nd;kd++) {
			v += data.contingence_var[kd*nvar*data.npos+ivar]*M[ikd*data.nd+kd];
		}

		for (int jniv=0;jniv<(nvar-iniv);jniv++) {
			/* fin entite constante..... */
			DT v2 =  v*data.contingence_var[ikd*nvar*data.npos+(ivar+jniv)];
			XX[GETACC(data,0,ipos,data.constCorrIndexCol[nfix + jniv + iniv],data.constCorrIndexCol[nfix + iniv])] += v2;
		}
	}
}


void QTLMapHomoscedasticAnimalModelCalcul::calcul_XT_X_C(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {

	dim3 dimBlock(BLOCKDIMX);
	dim3 dimGrid2 (ceil(data.npos*data.nLevelVar / dimBlock.x) + 1 , 1 );

	// initialize memory to 0
	XT_X_C_homoscedastic_animal_init<<<dimGrid2,dimBlock,0,stream>>>(data,work.XX);
	hipStreamSynchronize(stream);

	/* for each ikd we compute the following sub value  : IVAR[ .... ]    * M[ IKD, ....]   * CONT_VAR[....,IVAR:NVAR] */
	for (int ikd=0;ikd < data.nd; ikd++) {
		XT_X_C_homoscedastic_animal<<<dimGrid2,dimBlock,0,stream>>>(data,ikd,M,work.XX);
		hipStreamSynchronize(stream);
	}
}


__global__ void XT_X_C_heteroscedastic(DT * varInDevice,QTLMapStructDeviceDataLinear data,DT * XX) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;

	/* ivar : la diagonale de XT . X */

	if ( ivar < constNbLevelVar*data.npos && isim  < data.nsim ) {	           
		/* iniv position entre 0 et ntlevVar */ 
		size_t nfix = constNbLevelFix ;
		size_t nvar = constNbLevelVar ;

		/* niveau à une position */
		size_t iniv = ivar % nvar ;
		/* position teste */
		size_t ipos = ivar / nvar ;

		/* initialisation */
		/* On ne passse plus par un tableau local==> trop gourmand en moire du coup les grosse analyses ne passait pas (2QTL sur puce 54K)*
		   On accede directement a la globalMemory
		 */
		/* DT viniv[MAXND_WORK] ; */
		DT v=0.0,v2=0.0;

		//int scalesimXX = isim*BASE_SCALE_SYMMAT;
		v=0.0;
		for (size_t kd=0;kd<data.nd;kd++ ) {
			/* viniv[kd] = data.contingence_var[kd*nvar*data.npos+ivar];*/
			//v += viniv[kd]*viniv[kd]*(data.CD[kd] / varInDevice[ipos+isim*data.npos+data.corIpKd[kd]*data.nsim*data.npos]);
			v2 = data.contingence_var[kd*nvar*data.npos+ivar];
			v += v2*v2*(data.CD[kd] / varInDevice[ipos+isim*data.npos+data.corIpKd[kd]*data.nsim*data.npos]);	
		}		

		/* calcul de la diagonal */

		/* les diagonales sont stockees en premieres...*/ 
		XX[GETACC(data,isim,ipos,data.constCorrIndexCol[nfix + iniv],data.constCorrIndexCol[nfix + iniv])] = v;

		for (size_t jniv=1;jniv<(nvar-iniv);jniv++) {
			//scale +=  nvar*data.npos;
			DT v = 0.0;	

			for (size_t kd=0;kd<data.nd;kd++ ) {
				//v +=  viniv[kd]*data.contingence_var[kd*nvar*data.npos+(ivar+jniv)]*(data.CD[kd] / varInDevice[ipos+isim*data.npos+data.corIpKd[kd]*data.nsim*data.npos]);
				v2 = data.contingence_var[kd*nvar*data.npos+ivar];
				v +=  v2*data.contingence_var[kd*nvar*data.npos+(ivar+jniv)]*(data.CD[kd] / varInDevice[ipos+isim*data.npos+data.corIpKd[kd]*data.nsim*data.npos]);
			}
			XX[GETACC(data,isim,ipos,data.constCorrIndexCol[nfix + jniv + iniv],data.constCorrIndexCol[nfix + iniv])] = v;
		}
	}
}

__global__ void XT_X_C_heteroscedastic_opt_init(QTLMapStructDeviceDataLinear data,DT * XX) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;

	if ( ivar < constNbLevelVar*data.npos && isim  < data.nsim ) {	           
		/* iniv position entre 0 et ntlevVar */ 
		int nfix = constNbLevelFix ;
		int nvar = constNbLevelVar ;

		/* niveau à une position */
		int iniv = ivar % nvar ;
		/* position teste */
		int ipos = ivar / nvar ;

		for (int jniv=0;jniv<(nvar-iniv);jniv++) {	
			XX[GETACC(data,isim,ipos,data.constCorrIndexCol[nfix + jniv + iniv],data.constCorrIndexCol[nfix + iniv])] = 0;
		}
	}
}

__global__ void XT_X_C_heteroscedastic_opt(DT * varInDevice,int kd,QTLMapStructDeviceDataLinear data,DT * XX) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;

	/* ivar : la diagonale de XT . X */

	if ( ivar < constNbLevelVar*data.npos && isim  < data.nsim ) {	           
		/* iniv position entre 0 et ntlevVar */ 
		int nfix = constNbLevelFix ;
		int nvar = constNbLevelVar ;

		/* niveau à une position */
		int iniv = ivar % nvar ;
		/* position teste */
		int ipos = ivar / nvar ;

		DT v = data.contingence_var[kd*nvar*data.npos+ivar] ;
		for (int jniv=0;jniv<(nvar-iniv);jniv++) {	
			DT v2 =  v*data.contingence_var[kd*nvar*data.npos+(ivar+jniv)] *(data.CD[kd] / varInDevice[ipos+isim*data.npos+data.corIpKd[kd]*data.nsim*data.npos]);
			XX[GETACC(data,isim,ipos,data.constCorrIndexCol[nfix + jniv + iniv],data.constCorrIndexCol[nfix + iniv])] += v2;
		}
	}
}




void QTLMapHeteroscedasticModelCalcul::calcul_XT_X_C(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {
	int nbBlockY=1;
	while ( work.prop.maxGridSize[1] < ceil(data.nsim / nbBlockY )+1 ) {
		nbBlockY++;
	}

	dim3 dimBlock(BLOCKDIMX,nbBlockY);

	if ( work.prop.maxGridSize[0] < ceil(data.npos*data.nLevelVar / dimBlock.x) + 1 ) {
		cerr << "QTLMap can not support this number of position ["<< data.npos << "] to tested with the number of lever ["<< data.nLevelVar <<"]" << endl ;
		exit(1);
	}

	dim3 dimGrid (ceil(data.npos*data.nLevelVar / dimBlock.x) + 1 , ceil(data.nsim / nbBlockY )+1);
	XT_X_C_heteroscedastic<<<dimGrid,dimBlock,0,stream>>>(varInDevice,data,work.XX);
	//	// initialize memory to 0
	//	XT_X_C_heteroscedastic_opt_init<<<dimGrid,dimBlock,0,stream>>>(data,work.XX);
	//	hipStreamSynchronize(stream);
	//
	//	/* for each ikd we compute the following sub value  : IVAR[ .... ]    * M[ IKD, ....]   * CONT_VAR[....,IVAR:NVAR] */
	//	for (int ikd=0;ikd < data.nd; ikd++) {
	//		XT_X_C_heteroscedastic_opt<<<dimGrid,dimBlock,0,stream>>>(varInDevice,ikd,data,work.XX);
	//		hipStreamSynchronize(stream);
	//	}

}


/**
 *
 * Cholesky_Decomposition
 *
 *
 *
 */
__global__ void Cholesky_Decomposition_homoscedastic(QTLMapStructDeviceDataLinear data,DT * A_res,DT * XX, DT * triang) {

	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */

	//DT * triang = work.triang ;
	int ntniv = data.ntniv;
	int nfix = constNbLevelFix;

	if ( ipos < data.npos ) {

		for (int i=0;i<ntniv;i++) {
			for (int j=i;j<ntniv;j++) {
				triang[GETACC(data,0,ipos,i,j)] = 0.0;
			}
		}
		/* a optimiser , ces valeurs sont redondantes dans les matrices XX pour chaque position  */
		for (int i=0;i<nfix;i++) {
			for (int j=i;j<nfix;j++) {
				XX[GETACC(data,0,ipos,data.constCorrIndexCol[j],data.constCorrIndexCol[i])] = A_res[j*nfix+i];
			}
		}

		for (int j=0;j<ntniv;j++) {
			DT v = XX[GETACC(data,0,ipos,j,j)]; //XX[j][j] ;

			for (int k=0;k<=j-1;k++) {
				// [j][j] = [j][j] - [k][j]*[k][j]
				//v -= triang[ipos+k*data.npos+j*ntniv*data.npos]*triang[ipos+k*data.npos+j*ntniv*data.npos];
				v -= triang[GETACC(data,0,ipos,k,j)]*triang[GETACC(data,0,ipos,k,j)];
			}
			//si estimable .. on ne met pas de structure vecseuil==> on utilise la diagonale de triangle pour savoir l effet est estimable
			//triang[ipos+j*data.npos +j*ntniv*data.npos] =  sqrt(v);
			triang[GETACC(data,0,ipos,j,j)] =  sqrt(v);
			//if ( triang[ipos+j*data.npos +j*ntniv*data.npos] != triang[ipos+j*data.npos +j*ntniv*data.npos] ) triang[ipos+j*data.npos +j*ntniv*data.npos] = 0.0;
			if ( triang[GETACC(data,0,ipos,j,j)] != triang[GETACC(data,0,ipos,j,j)] ) triang[GETACC(data,0,ipos,j,j)] = 0.0;
			if ( triang[GETACC(data,0,ipos,j,j)] > data.seuil_cho ) { 
				for (int i=j+1;i<ntniv;i++ ) {
					//triang[ipos+i*data.npos +j*ntniv*data.npos] = work.XX[ipos+i*data.npos +j*ntniv*data.npos];//XX[i][j];
					triang[GETACC(data,0,ipos,i,j)] = XX[GETACC(data,0,ipos,i,j)];//XX[i][j];
					for ( int k=0;k<=j-1;k++) {
						triang[GETACC(data,0,ipos,i,j)] -= (triang[GETACC(data,0,ipos,i,k)] * triang[GETACC(data,0,ipos,j,k)]);
					}
					triang[GETACC(data,0,ipos,i,j)] = triang[GETACC(data,0,ipos,i,j)] / triang[GETACC(data,0,ipos,j,j)];
					//triang[ipos+j*data.npos +i*ntniv*data.npos] = triang[ipos+i*data.npos +j*ntniv*data.npos] ;
				}
			}  
		}
	}

}


void QTLMapHomoscedasticModelCalcul::calcul_Cholesky_Decomposition(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {

	int nbblock_512 = ceil(data.npos / MAX_BLOCKDIM_512) + 1 ;
	Cholesky_Decomposition_homoscedastic<<<nbblock_512,MAX_BLOCKDIM_512,0,stream>>>(data,work.A_res,work.XX,work.triang);
}


__global__ void Cholesky_Decomposition_heterocedastic(QTLMapStructDeviceDataLinear data,DT * XX, DT * triang) {

	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;

	//DT * triang = work.triang ;
	int ntniv = data.ntniv;

	if ( ipos < data.npos && isim < data.nsim ) {
		for (int i=0;i<ntniv;i++) {
			for (int j=i;j<ntniv;j++) {
				triang[GETACC(data,isim,ipos,i,j)] = 0.0;
			}
		}


		for (int j=0;j<ntniv;j++) {
			DT v = XX[GETACC(data,isim,ipos,j,j)]; //XX[j][j] ;

			for (int k=0;k<=j-1;k++) {
				// [j][j] = [j][j] - [k][j]*[k][j]
				v -= triang[GETACC(data,isim,ipos,j,k)]*triang[GETACC(data,isim,ipos,j,k)];
			}

			//si estimable .. on ne met pas de structure vecseuil==> on utilise la diagonale de triangle pour savoir l effet est estimable
			triang[GETACC(data,isim,ipos,j,j)] =  sqrt(v);
			if ( triang[GETACC(data,isim,ipos,j,j)] != triang[GETACC(data,isim,ipos,j,j)] ) triang[GETACC(data,isim,ipos,j,j)] = 0.0;

			if ( triang[GETACC(data,isim,ipos,j,j)] > data.seuil_cho ) { 
				for (int i=j+1;i<ntniv;i++ ) {
					triang[GETACC(data,isim,ipos,i,j)] = XX[GETACC(data,isim,ipos,i,j)];//XX[i][j];
					for ( int k=0;k<=j-1;k++) {
						triang[GETACC(data,isim,ipos,i,j)] -= (triang[GETACC(data,isim,ipos,i,k)] * triang[GETACC(data,isim,ipos,j,k)]);
					}
					triang[GETACC(data,isim,ipos,i,j)] = triang[GETACC(data,isim,ipos,i,j)] / triang[GETACC(data,isim,ipos,j,j)];
				}
			}
		}
	}

}


void QTLMapHeteroscedasticModelCalcul::calcul_Cholesky_Decomposition(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {
	int nbBlockY=1;
	while ( work.prop.maxGridSize[1] < ceil(data.nsim / nbBlockY )+1 ) {
		nbBlockY++;
	}

	dim3 dimBlock(BLOCKDIMX,nbBlockY);

	if ( work.prop.maxGridSize[0] < ceil(data.npos / dimBlock.x) + 1 ) {
		cerr << "QTLMap can not support this number of position ["<< data.npos << "] to tested."<< endl ;
		exit(1);
	}

	dim3 dimGrid (ceil(data.npos / dimBlock.x) + 1 , ceil(data.nsim / nbBlockY )+1);
	Cholesky_Decomposition_heterocedastic<<<dimGrid,dimBlock,0,stream>>>(data,work.XX,work.triang);
}

/*
 * A optimiser : rhs pour les effet fixe a la position ne devrait prendre que nsim*nlevelfix et non pas nsim*nlevelfix*npos
 * 
 * 
 * logique de corrIndexCol :
 * 
 *     I=1,..,NTNIV | I=1,..,NFIX,NFIX+1,..,NVAR
 *     -------------------------------------------
 *     0            |            0
 *     1            |            J
 *     2            |            J+1
 *     3            |            .
 *     .            |            1
 *     .            |            2
 *     .            |            3
 *     NTNIV        |            NTNIV
 * 
 * 
 * 
 */
__global__ void Set_RHS_FIX_homoscedatic(QTLMapStructDeviceDataLinear data,DT * rhs,DT * triang) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;// * blockDim.y + threadIdx.y ;

	int idNiv;
	if ( ipos < data.npos && isim < data.nsim ) {
		for (int iniv=0;iniv<constNbLevelFix;iniv++) {

			idNiv = data.constCorrIndexCol[iniv];

			int index = ipos + isim*data.npos + idNiv*data.nsim*data.npos ;
			DT v=0;
			if (triang[GETACC(data,0,ipos,idNiv,idNiv)]>data.seuil_cho) {	
				for (int kd = 0;kd<data.nd;kd++ ) {
					v += data.contingence_fix[iniv*data.ndmax + kd]*data.Y[isim+kd*data.nsim];
				}	
				rhs[index] = v;
			}
		}
	}
}

__global__ void Set_RHS_VAR_homoscedatic(QTLMapStructDeviceDataLinear data,DT * rhs,DT * triang) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y ;// * blockDim.y + threadIdx.y ;

	if ( ivar < constNbLevelVar*data.npos ) {	
		/* iniv position entre 0 et ntlevVar */ 

		/* niveau à une position */
		int iniv = ivar % constNbLevelVar ;
		/* position teste */
		int ipos = ivar / constNbLevelVar ;

		//for (int isim=0;isim<data.nsim;isim++) {
		int index = ipos + isim*data.npos + data.constCorrIndexCol[constNbLevelFix+iniv]*data.nsim*data.npos ;
		DT v=0;
		if (triang[GETACC(data,0,ipos,data.constCorrIndexCol[constNbLevelFix+iniv],data.constCorrIndexCol[constNbLevelFix+iniv])]>data.seuil_cho) {
			for (int kd = 0;kd<data.nd;kd++ ) {
				v += data.contingence_var[kd*constNbLevelVar*data.npos+ivar]*data.Y[isim+kd*data.nsim];
			}
			rhs[index] = v;
		}
		//}
	}
}


void QTLMapHomoscedasticModelCalcul::calcul_RHS(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {
	dim3 dimBlock_Set_RHS_FIX(16,32);
	dim3 dimGrid_Set_RHS_FIX (ceil(data.npos*data.nLevelVar /dimBlock_Set_RHS_FIX.x ) + 1 ,ceil(data.nsim / dimBlock_Set_RHS_FIX.y )+1);
	Set_RHS_FIX_homoscedatic<<<dimGrid_Set_RHS_FIX,dimBlock_Set_RHS_FIX,0,stream>>>(data,work.rhs,work.triang);
	dim3 dimGrid_Set_RHS_VAR (ceil(data.npos*data.nLevelVar / MAX_BLOCKDIM_64) + 1 ,data.nsim);
	Set_RHS_VAR_homoscedatic<<<dimGrid_Set_RHS_VAR,MAX_BLOCKDIM_64,0,stream>>>(data,work.rhs,work.triang);
}

//***********************************************************************************************************************************************

__global__ void Set_RHS_FIX_homoscedatic_animal(QTLMapStructDeviceDataLinear data,DT *FixM,DT * M,DT * rhs,DT * triang) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;// * blockDim.y + threadIdx.y ;

	int idNiv;
	if ( ipos < data.npos && isim < data.nsim ) {
		for (int iniv=0;iniv<constNbLevelFix;iniv++) {

			idNiv = data.constCorrIndexCol[iniv];
			int index = ipos + isim*data.npos + idNiv*data.nsim*data.npos ;
			DT v2=0;
			if (triang[GETACC(data,0,ipos,idNiv,idNiv)]>data.seuil_cho) {	
				for (int kd = 0;kd<data.nd;kd++ ) {
					v2 += FixM[kd*constNbLevelFix+iniv]*data.Y[isim+kd*data.nsim];
				}	
				rhs[index] = v2 ;
			}
		}
	}
}

__global__ void Set_RHS_VAR_homoscedatic_animal_1(QTLMapStructDeviceDataLinear data,DT * M,DT * tempRHS) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ;
	int kd   = blockIdx.y ;

	if ( ivar < constNbLevelVar*data.npos && kd < data.nd ) {
		DT v=0.0;
		for (int ikd=0;ikd<data.nd;ikd++) {
			v += data.contingence_var[ikd*constNbLevelVar*data.npos+ivar] * M[ikd*data.nd+kd];
		}
		tempRHS[kd*constNbLevelVar*data.npos + ivar] = v;
	}			
}

__global__ void Set_RHS_VAR_homoscedatic_animal_2(QTLMapStructDeviceDataLinear data,DT * tempRHS,DT * rhs,DT * triang) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y ;// * blockDim.y + threadIdx.y ;

	if ( ivar < constNbLevelVar*data.npos && isim < data.nsim ) {	
		/* iniv position entre 0 et ntlevVar */ 

		/* niveau à une position */
		int iniv = ivar % constNbLevelVar ;
		/* position teste */
		int ipos = ivar / constNbLevelVar ;

		//for (int isim=0;isim<data.nsim;isim++) {
		int index = ipos + isim*data.npos + data.constCorrIndexCol[constNbLevelFix+iniv]*data.nsim*data.npos ;
		DT v=0.0;
		if (triang[GETACC(data,0,ipos,data.constCorrIndexCol[constNbLevelFix+iniv],data.constCorrIndexCol[constNbLevelFix+iniv])]>data.seuil_cho) {
			for (int kd = 0;kd<data.nd;kd++ ) {
				v += tempRHS[kd*constNbLevelVar*data.npos+ivar]*data.Y[isim+kd*data.nsim];
			}
			rhs[index] = v;
		}
		//}
	}
}


void QTLMapHomoscedasticAnimalModelCalcul::calcul_RHS(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {
	dim3 dimBlock_Set_RHS_FIX(16,32);
	dim3 dimGrid_Set_RHS_FIX (ceil(data.npos*data.nLevelVar /dimBlock_Set_RHS_FIX.x ) + 1 ,ceil(data.nsim / dimBlock_Set_RHS_FIX.y )+1);
	Set_RHS_FIX_homoscedatic_animal<<<dimGrid_Set_RHS_FIX,dimBlock_Set_RHS_FIX,0,stream>>>(data,FixM,M,work.rhs,work.triang);

	dim3 dimBlock_1(MAX_BLOCKDIM_64);
	dim3 dimGrid_1 (ceil(data.npos*data.nLevelVar / dimBlock_1.x) + 1 ,ceil(data.nd / dimBlock_1.y )+1);
	size_t size = data.nLevelVar*data.npos*data.nd*sizeof(DT);
	DT * tempRHS = NULL ;
	safecall(hipMalloc(&tempRHS,size));
	Set_RHS_VAR_homoscedatic_animal_1<<<dimGrid_1,dimBlock_1,0,stream>>>(data,M,tempRHS); 

	dim3 dimBlock_2(MAX_BLOCKDIM_64);
	dim3 dimGrid_2(ceil(data.npos*data.nLevelVar / dimBlock_2.x) + 1 ,ceil(data.nsim / dimBlock_2.y )+1);
	Set_RHS_VAR_homoscedatic_animal_2<<<dimGrid_2,dimBlock_2,0,stream>>>(data,tempRHS,work.rhs,work.triang);
	safecall(hipFree(tempRHS));

}


//***********************************************************************************************************************************************

__global__ void Set_RHS_FIX_heteroscedastic(DT * varInDevice,QTLMapStructDeviceDataLinear data,DT * rhs,DT * triang) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;// * blockDim.y + threadIdx.y ;

	int idNiv;
	if ( ipos < data.npos && isim < data.nsim ) {
		for (int iniv=0;iniv<constNbLevelFix;iniv++) {

			idNiv = data.constCorrIndexCol[iniv];

			int index = ipos + isim*data.npos + idNiv*data.nsim*data.npos ;
			rhs[index]=0;
			if (triang[GETACC(data,isim,ipos,idNiv,idNiv)]>data.seuil_cho) {
				for (int kd=0;kd<data.nd;kd++ ) {
					rhs[index] += data.contingence_fix[iniv*data.ndmax + kd]*data.Y[isim+kd*data.nsim]*(data.CD[kd] / varInDevice[ipos+isim*data.npos+data.corIpKd[kd]*data.nsim*data.npos]);
				}
			}
		}
	}
}

__global__ void Set_RHS_VAR_heteroscedastic(DT * varInDevice,QTLMapStructDeviceDataLinear data,DT * rhs,DT * triang) {
	int ivar = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;// * blockDim.y + threadIdx.y ;

	if ( ivar < constNbLevelVar*data.npos && isim < data.nsim ) {	
		/* iniv position entre 0 et ntlevVar */ 

		/* niveau à une position */
		int iniv = ivar % constNbLevelVar ;
		/* position teste */
		int ipos = ivar / constNbLevelVar ;

		int index = ipos + isim*data.npos + data.constCorrIndexCol[constNbLevelFix+iniv]*data.nsim*data.npos ;

		rhs[index]=0;

		if (triang[GETACC(data,isim,ipos,data.constCorrIndexCol[constNbLevelFix+iniv],data.constCorrIndexCol[constNbLevelFix+iniv])]>data.seuil_cho) {
			for (int kd = 0;kd<data.nd;kd++ ) {
				rhs[index] += data.contingence_var[kd*constNbLevelVar*data.npos+ivar]*data.Y[isim+kd*data.nsim]*(data.CD[kd] / varInDevice[ipos+isim*data.npos+data.corIpKd[kd]*data.nsim*data.npos]);
			}
		}
	}
}

void QTLMapHeteroscedasticModelCalcul::calcul_RHS(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work) {
	dim3 dimBlock_Set_RHS_FIX(16,32);
	dim3 dimGrid_Set_RHS_FIX (ceil(data.npos*data.nLevelVar /dimBlock_Set_RHS_FIX.x ) + 1 ,ceil(data.nsim / dimBlock_Set_RHS_FIX.y )+1);
	Set_RHS_FIX_heteroscedastic<<<dimGrid_Set_RHS_FIX,dimBlock_Set_RHS_FIX,0,stream>>>(varInDevice,data,work.rhs,work.triang);

	int nbBlockY=1;
	while ( work.prop.maxGridSize[1] < ceil(data.nsim / nbBlockY )+1 ) {
		nbBlockY++;
	}
	dim3 dimBlock(MAX_BLOCKDIM_64,nbBlockY);
	dim3 dimGrid_Set_RHS_VAR (ceil(data.npos*data.nLevelVar / MAX_BLOCKDIM_64) + 1 ,ceil(data.nsim / nbBlockY )+1);
	Set_RHS_VAR_heteroscedastic<<<dimGrid_Set_RHS_VAR,dimBlock,0,stream>>>(varInDevice,data,work.rhs,work.triang);
}

/**
 *
 * Resolution LU (Partie descendante descendante )
 *
 *
 */
__global__ void Resolve_LU_homoscedastic(QTLMapStructDeviceDataLinear data, DT * rhs, DT * triang,DT * bestim) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;

	int ntniv = data.ntniv ;
	DT temp[MAXNTNIV_LOCALTHREAD_MEMORY];

	/******* DESCENT LU *****/
	if ( ipos < data.npos && isim < data.nsim ) {
		//for (int isim=0;isim<data.nsim;isim++) {
		int scale = ipos + isim*data.npos ;
		for (int i=0;i<ntniv;i++) {
			temp[i] = 0.0;
			if (triang[GETACC(data,0,ipos,i,i)]>data.seuil_cho) {
				temp[i] = rhs[scale + i*data.nsim*data.npos];
				for (int j=i-1;j>=0;j--) {
					if (triang[GETACC(data,0,ipos,j,j)]>data.seuil_cho) {
						temp[i] = temp[i] - temp[j]*triang[GETACC(data,0,ipos,i,j)];
					}
				}
				temp[i] = temp[i] / triang[GETACC(data,0,ipos,i,i)];
			} 
		}

		for (int i=ntniv-1;i>=0;i--) {
			int indexI = scale + i*data.nsim*data.npos;
			bestim[indexI] = 0.0 ;
			if (triang[GETACC(data,0,ipos,i,i)]>data.seuil_cho) {
				bestim[indexI] = temp[i];
				for (int j=i+1;j<ntniv;j++) {
					if (triang[GETACC(data,0,ipos,j,j)]>data.seuil_cho) {
						int indexJ = scale + j*data.nsim*data.npos;
						bestim[indexI] -= bestim[indexJ]*triang[GETACC(data,0,ipos,i,j)];
					}
				}
				bestim[indexI] = bestim[indexI] / triang[GETACC(data,0,ipos,i,i)]; 
			} 
		}
		//}//fin for isim

	}
}


void QTLMapHomoscedasticModelCalcul::calcul_LU(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) {
	/* Solution */
	dim3 Resolve_LU_dimBlock(16,32);
	dim3 Resolve_LU_dimGrid (ceil(data.npos/Resolve_LU_dimBlock.x)+1 , ceil(data.nsim / Resolve_LU_dimBlock.y)+1);
	Resolve_LU_homoscedastic<<<Resolve_LU_dimGrid,Resolve_LU_dimBlock,0,stream>>>(data,work.rhs,work.triang,solution.bestim);
}


__global__ void Resolve_LU_heteroscedastic(QTLMapStructDeviceDataLinear data, DT * rhs, DT * triang,DT * bestim) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ;

	int ntniv = data.ntniv ;

	DT temp[MAXNTNIV_LOCALTHREAD_MEMORY];

	/******* DESCENT LU *****/
	if ( ipos < data.npos && isim < data.nsim ) {
		int scale = ipos + isim*data.npos ;
		for (int i=0;i<ntniv;i++) {
			temp[i] = 0.0;
			if (triang[GETACC(data,isim,ipos,i,i)]>data.seuil_cho) {
				temp[i] = rhs[scale + i*data.nsim*data.npos];
				for (int j=i-1;j>=0;j--) {
					if (triang[GETACC(data,isim,ipos,j,j)]>data.seuil_cho) {
						temp[i] = temp[i] - temp[j]*triang[GETACC(data,isim,ipos,j,i)];
					}
				}
				temp[i] = temp[i] / triang[GETACC(data,isim,ipos,i,i)];
			} 
		}

		for (int i=ntniv-1;i>=0;i--) {
			int indexI = scale + i*data.nsim*data.npos;
			bestim[indexI] = 0.0 ;
			if (triang[GETACC(data,isim,ipos,i,i)]>data.seuil_cho) {
				bestim[indexI] = temp[i];
				for (int j=i+1;j<ntniv;j++) {
					if (triang[GETACC(data,isim,ipos,j,j)]>data.seuil_cho) {
						int indexJ = scale + j*data.nsim*data.npos;
						bestim[indexI] -= bestim[indexJ]*triang[GETACC(data,isim,ipos,i,j)];
					}
				}
				bestim[indexI] = bestim[indexI] / triang[GETACC(data,isim,ipos,i,i)]; 
			} 
		}
	}
}



void QTLMapHeteroscedasticModelCalcul::calcul_LU(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) {
	/* Solution */
	dim3 Resolve_LU_dimBlock(16,32);
	dim3 Resolve_LU_dimGrid (ceil(data.npos/Resolve_LU_dimBlock.x)+1 , ceil(data.nsim / Resolve_LU_dimBlock.y)+1);
	Resolve_LU_heteroscedastic<<<Resolve_LU_dimGrid,Resolve_LU_dimBlock,0,stream>>>(data,work.rhs,work.triang,solution.bestim);
}




/**
 *
 * Set_SIGSQ
 * XB = Y - ( X' . Bestim )
 * SIGSQ = SUM(XB^2)
 */
__global__ void Set_SIGSQ_homoscedastic(QTLMapStructDeviceDataLinear data,DT * bestim,DT * osigsq) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ; /* simulation */

	int ntniv = data.ntniv ;

	if ( ipos < data.npos && isim < data.nsim ) {
		//	for (int isim=0;isim<data.nsim;isim++) {
		int scale = ipos + isim*data.npos ;
		DT xb = 0 ;

		for (int kd = 0;kd<data.nd;kd++ ) {
			DT v=0;

#pragma unroll 4
			for (int iniv=0;iniv<constNbLevelFix;iniv++) {
				v += data.contingence_fix[iniv*data.ndmax+kd]*bestim[scale + data.constCorrIndexCol[iniv]*data.nsim*data.npos];
			}

			for (int iniv=constNbLevelFix;iniv<ntniv;iniv++) {	
				v += data.contingence_var[kd*constNbLevelVar*data.npos+ipos*constNbLevelVar+(iniv-constNbLevelFix)]*bestim[scale + data.constCorrIndexCol[iniv]*data.nsim*data.npos];
			}

			v = data.Y[isim+kd*data.nsim] - v;
			xb += v*v ;
		}
		DT v = xb / data.nd ;

#pragma unroll 4
		for (int ip=0;ip<data.np;ip++) {
			osigsq[ipos+isim*data.npos+ip*data.npos*data.nsim] = v ; 
		}
	}
	//}
}

void QTLMapHomoscedasticModelCalcul::calcul_SIGSQ(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) {
	dim3 dimBlock(16,32);
	dim3 dimGrid (ceil(data.npos/dimBlock.x)+1 , ceil(data.nsim / dimBlock.y)+1);					
	Set_SIGSQ_homoscedastic<<<dimGrid,dimBlock,0,stream>>>(data,solution.bestim,solution.osigsq);
}
//****************************************************************************************************************************************************
__global__ void Set_SIGSQ_homoscedastic_animal(QTLMapStructDeviceDataLinear data,DT * M,DT * bestim,DT * osigsq) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ; /* simulation */

	int ntniv = data.ntniv ;

	if ( ipos < data.npos && isim < data.nsim ) {
		//	for (int isim=0;isim<data.nsim;isim++) {
		int scale = ipos + isim*data.npos ;
		DT xb[5000]  ; /* attention ce 5000 remplace MAXND qui n est plus utilisé....ceci est pour l'experimentation du modele animale....*/

		for (int kd = 0;kd<data.nd;kd++ ) {
			DT v=0;

#pragma unroll 4
			for (int iniv=0;iniv<constNbLevelFix;iniv++) {
				v += data.contingence_fix[iniv*data.ndmax+kd]*bestim[scale + data.constCorrIndexCol[iniv]*data.nsim*data.npos];
			}

			for (int iniv=constNbLevelFix;iniv<ntniv;iniv++) {	
				v += data.contingence_var[kd*constNbLevelVar*data.npos+ipos*constNbLevelVar+(iniv-constNbLevelFix)]*bestim[scale + data.constCorrIndexCol[iniv]*data.nsim*data.npos];
			}

			v = data.Y[isim+kd*data.nsim] - v;
			xb[kd] = v;
		}

		DT tot=0;
		for ( int kd=0;kd<data.nd;kd++  ) {
			DT v=0;
			for (int kd2 = 0;kd2<data.nd;kd2++ ) {
				v += xb[kd2]*M[kd*data.nd+kd2];
			}
			tot += v*xb[kd];
		}

		DT v = tot / data.nd ;

#pragma unroll 4
		for (int ip=0;ip<data.np;ip++) {
			osigsq[ipos+isim*data.npos+ip*data.npos*data.nsim] = v ; 
			//osigsq[ipos+isim*data.npos+ip*data.npos*data.nsim] = 0.5;
		}

	}
	//}
}


//****************************************************************************************************************************************************
__global__ void Set_SIGSQ_homoscedastic_animal_XB(QTLMapStructDeviceDataLinear data,DT * XB,DT * bestim) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ; /* simulation */

	int ntniv = data.ntniv ;

	if ( ipos < data.npos && isim < data.nsim ) {
		int scale = ipos + isim*data.npos ;

		for (int kd = 0;kd<data.nd;kd++ ) {
			DT v=0;

			for (int iniv=0;iniv<constNbLevelFix;iniv++) {
				v += data.contingence_fix[iniv*data.ndmax+kd]*bestim[scale + data.constCorrIndexCol[iniv]*data.nsim*data.npos];
			}

			for (int iniv=constNbLevelFix;iniv<ntniv;iniv++) {	
				v += data.contingence_var[kd*constNbLevelVar*data.npos+ipos*constNbLevelVar+(iniv-constNbLevelFix)]*bestim[scale + data.constCorrIndexCol[iniv]*data.nsim*data.npos];
			}

			XB[kd*data.nsim*data.npos+scale] = data.Y[isim+kd*data.nsim] - v;
		}		
	}
}

__global__ void Set_SIGSQ_homoscedastic_animal_2(QTLMapStructDeviceDataLinear data,DT * M,DT * XB,DT * osigsq) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y ; /* simulation */

	if ( ipos < data.npos && isim < data.nsim ) {
		int scale = ipos + isim*data.npos ;

		DT tot=0;
		for ( int kd=0;kd<data.nd;kd++  ) {
			DT v=0;
			for (int kd2 = 0;kd2<data.nd;kd2++ ) {
				v += XB[kd2*data.nsim*data.npos+scale]*M[kd*data.nd+kd2];
				//v += xb_loc[kd2]*M[kd*data.nd+kd2];
			}
			tot += v*XB[kd*data.nsim*data.npos+scale];
			//		tot += v*xb_loc[kd];
		}

		DT v = tot / data.nd ;

		for (int ip=0;ip<data.np;ip++) {
			osigsq[ipos+isim*data.npos+ip*data.npos*data.nsim] = v ; 
		}	
	}
}

void QTLMapHomoscedasticAnimalModelCalcul::calcul_SIGSQ(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) {
	dim3 dimBlock(16,32);
	dim3 dimGrid (ceil(data.npos/dimBlock.x)+1 , ceil(data.nsim / dimBlock.y)+1);	
	Set_SIGSQ_homoscedastic_animal<<<dimGrid,dimBlock,0,stream>>>(data,M,solution.bestim,solution.osigsq);

	//	int size=data.npos*data.nsim*data.nd*sizeof(DT);
	//	DT * XB = NULL ;
	//	safecall(hipMalloc(&XB,size));
	//
	//	dim3 dimBlock(32,32);
	//	dim3 dimGrid (ceil(data.npos/dimBlock.x)+1 , ceil(data.nsim / dimBlock.y)+1);
	//
	//	Set_SIGSQ_homoscedastic_animal_XB<<<dimGrid,dimBlock,0,stream>>>(data,XB,solution.bestim);
	//
	//	Set_SIGSQ_homoscedastic_animal_2<<<dimGrid,dimBlock,0,stream>>>(data,M,XB,solution.osigsq);
	//	safecall(hipFree(XB));

}
//****************************************************************************************************************************************************
/**
 *
 * Set_SIGSQ
 * XB = Y - ( X' . Bestim )
 * SIGSQ = SUM(XB^2)
 */
__global__ void Set_SIGSQ_heteroscedastic(QTLMapStructDeviceDataLinear data,DT * bestim,DT * osigsq) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int isim = blockIdx.y * blockDim.y + threadIdx.y  ; 

	int ntniv = data.ntniv ;

	if ( ipos < data.npos && isim < data.nsim ) {
		int scale = ipos + isim*data.npos ;

		for (int ip=0;ip<data.np;ip++) {
			DT xb = 0 ; 
			for (int kd = 0;kd<data.nd;kd++ ) {
				if ( data.corIpKd[kd] == ip ) {
					DT v=0;

					for (int iniv=0;iniv<constNbLevelFix;iniv++) {
						v += data.contingence_fix[iniv*data.ndmax+kd]*bestim[scale + data.constCorrIndexCol[iniv]*data.nsim*data.npos];
					}

					for (int iniv=constNbLevelFix;iniv<ntniv;iniv++) {	
						v += data.contingence_var[kd*constNbLevelVar*data.npos+ipos*constNbLevelVar+(iniv-constNbLevelFix)]*bestim[scale + data.constCorrIndexCol[iniv]*data.nsim*data.npos];
					}

					v = data.Y[isim+kd*data.nsim] - v;
					xb += v*v*data.CD[kd]*data.CD[kd] ;
				}
			}
			osigsq[ipos+isim*data.npos+ip*data.npos*data.nsim] = xb / constSizeFam[ip];
		}
	}
}


void QTLMapHeteroscedasticModelCalcul::calcul_SIGSQ(hipStream_t stream,QTLMapStructDeviceDataLinear data,QTLMapStructDeviceWorkLinear work,QTLMapStructDeviceSolutionLinear solution) {
	dim3 dimBlock(16,32);
	dim3 dimGrid (ceil(data.npos/dimBlock.x)+1 , ceil(data.nsim / dimBlock.y)+1);			
	Set_SIGSQ_heteroscedastic<<<dimGrid,dimBlock,0,stream>>>(data,solution.bestim,solution.osigsq);
	safecall(hipGetLastError());
}

/**
 *
 *  Set_LRT
 *
 *  Modif Avril 2013 : return LRT sire contribution
 */
__global__ void Set_LRT(QTLMapStructDeviceDataLinear data,DT * osigsq, DT * lrt,int scalePosition) {
	int ipos = blockIdx.x * blockDim.x + threadIdx.x ; /* position */
	int ip   = blockIdx.y ;

	if ( data.nqtl > 0 ) {
		if ( ipos < data.npos && ip < data.np ) {
			for (int isim=0;isim<data.nsim;isim++) {

				int k = ipos + isim * data.npos ;

				for (int iqtl=0;iqtl<data.nqtl;iqtl++) {
					//					DT v = 0.0;
					//					for (int ip=0;ip<data.np;ip++) {
					//						v += constSizeFam[ip] * (log(data.isigsq[isim+ip*data.nsim+(iqtl)*data.nsim*data.np]) - log(osigsq[k+ip*data.npos*data.nsim]));
					//					}
					lrt[scalePosition+ipos+isim*data.nposGlobal+iqtl*data.nposGlobal*data.nsim+ip*data.nqtl*data.nposGlobal*data.nsim] =
							constSizeFam[ip] * (log(data.isigsq[isim+ip*data.nsim+(iqtl)*data.nsim*data.np]) - log(osigsq[k+ip*data.npos*data.nsim]));
				}
			}//fin for isim
		} 
	}
}

extern "C" void release_allocated_internal_sig_() 
{
	if (allVarFitted != NULL ) delete[] allVarFitted;
	allVarFitted = NULL;
}

/**
 * - save residual variance at the maximum position of the likelihood. this information is useful to compute LRT
 */
void add_internal_sig(DT * osigsquare,int *maxPosition,int npos,int nsim,int np,int nqtl) {
	DT * allVarFitted_buf = NULL ;
	if ( allVarFitted ) {
		allVarFitted_buf = new DT [nqtl*np*nsim];
		for (int iq=0;iq<nqtl;iq++) {
			for (int isim=0;isim<nsim;isim++) {
				for (int ip=0;ip<np;ip++)
					allVarFitted_buf[isim+ip*nsim+iq*np*nsim] = allVarFitted[isim+ip*nsim+iq*np*nsim];
			}
		}
		delete[] allVarFitted;
		allVarFitted = NULL;
	}

	allVarFitted = new DT [(nqtl+1)*np*nsim];
	if ( nqtl >= 1 ) {
		for (int iq=0;iq<nqtl;iq++) {
			for (int isim=0;isim<nsim;isim++) {
				for (int ip=0;ip<np;ip++) {
					allVarFitted[isim+ip*nsim+iq*np*nsim] = allVarFitted_buf[isim+ip*nsim+iq*np*nsim];
				}
			}
		}
	}

	if (allVarFitted_buf) {
		delete [] allVarFitted_buf ;
		allVarFitted_buf = NULL;
	}

	/* update with the new values osig computed */
	for (int isim=0;isim<nsim;isim++) {
		if ( maxPosition[isim] >= 1 ) { /* index fortran */
			int ipos = maxPosition[isim] - 1;
			for (int ip=0;ip<np;ip++) {
				//			cout << osigsquare[ipos+isim*npos+ip*nsim*npos] << endl ;
				allVarFitted[isim+ip*nsim+nqtl*np*nsim] = osigsquare[ipos+isim*npos+ip*nsim*npos];	
			}
		}
	}
}



/*
  xinc  = matrice d'incidence
  nkd   = nombre de ligne
  ntniv = nombre de colonne
  seuil_choPtr = seuil d'estimabilite pour la solution

  column major order ===> offset = row + column*NUMROWS
  XINC ACCESS => ndmax*npos*iniv+npos*kd+ipos

  XINC(construction : level fixe puis levels variables)
  Mu MuPere1 MuPere2 MuMere1 MuMere2 QTLPere1Pos1 QTLPere2Pos1 QTLMere1Pos1 QTLMere2Pos1 ... QTLPere1PosN QTLPere2PosN QTLMere1PosN QTLMere2PosN

  nLevelFixPtr : 5 (Moyenne gen + 4 effet polygenique)
  nLevelVarPtr : 4 (4 effet qtl a estimer)
  corrLevelColPtr(1) = 1    moyenne generale
  corrLevelColPtr(2) = 6    effet qtl pere 1
  corrLevelColPtr(3) = 7    effet qtl pere 2
  corrLevelColPtr(4) = 8    effet qtl mere 1
  corrLevelColPtr(5) = 9    effet qtl mere 2
  corrLevelColPtr(6) = 2    effet polygenique pere 1
  corrLevelColPtr(7) = 3    effet polygenique pere 2
  corrLevelColPtr(8) = 4    effet polygenique mere 1
  corrLevelColPtr(9) = 5    effet polygenique mere 2
 */
extern "C" void cuda_model_resolv_genome_(
		int *gpu_device_id,                   /* IN  : The id device to used */
		int *heteroscedastic_mode,            /* IN  : heteroscedastic = 1 , homoscedastic = 0,  homoscedastic_animal = 2 */
		int *nqtlPtr,                         /* IN  : Number of QTL */
		DT  *sigsquare,                       /* IN  : variance residuelle sous les hypothese 0 à NQTL-1 tableau [NSIM][NP] */
		int *nLevelFixPtr,                    /* IN  : nombre de niveaux fixe a la position : moyenne genrale, polygenic, effet fixe, covariables */
		int *nLevelVarPtr,                    /* IN  : nombre de niveau variable a la position */
		int *corrLevelColPtr,                 /* IN  : Correspondance du niveau associe a une colonne de la matrice de contingence */
		void *work_cuda,                      /* IN  : structure de travail pour la fonction get_partialXinc */
		void (*get_partialXincFix)(void*,DT*),   /* IN  : Fonction qui donne les donnees fixe a la position de la matrice d incidences... */
		void (*get_partialXincVar)(void*,int*,int*,DT*),            /* IN  : Fonction qui donne les donnees variables a la position des matrices d incidences... */
		DT *Y_d,                              /* IN  : performance [nsim,nd] */
		DT *CD_d,                             /* IN  : cd of anim  [nd]      */
		DT *M_d ,                             /* IN  : matrix quantity (I - (I -lambda. A ** -1))  [nd][nd]      */
		int *nsimPtr,                         /* IN  : nombre de simulation */
		int *ndPtr,                           /* IN  : nombre d'individu total  : nombre de ligne reelle de XINC */
		int *nkdPtr,                          /* IN  : nombre d'individu pris en compte dans les matrices de contingence */
		int *npPtr,                           /* IN  : nombre de famille de pere */
		int *sizeFamilyNp,                    /* IN  : nombre de descendant par pere tableau [NP] */
		int *ntnivmaxPtr,                     /* IN  : nombre maximum de niveau : nombre de colonne reelle de XINC */
		int *npositionPtr,                    /* IN  : number of position estimation */
		double  *seuil_choPtr,                /* IN  : seuil pour considere grace a la decomposition de cholesky si un niveau est estimable */
		//		int    * vecsol,                      /* OUT : tableau de booleen de taille [NPOSITION,NTNIV]  pour l'estimabilite de chaque niveau */
		DT * bestim,                          /* OUT : tableau solution de l'estimation de chaque niveau (taille [NPOSITION,NSIM,NTNIV]) */
		DT * osigsquare,                      /* OUT : tableau des variances residuelles pour chaque familles de pere (taille [NPOSITION,NSIM,NP])  */
		DT * lrt,                             /* OUT : LRT (taille [NPOSITION,NSIM,NQTL,NP])  */
		DT * maxLRT,                          /* OUT : Value of the Maximul LRT  [NQTL,NSIM] */
		int *maxPosition                      /* OUT : Index of the position maximum tableau [NSIM] */
) {
	/* reset the cards to remove leaks */
	safecall(hipDeviceReset());
#if(_CUDA_HOST_DEBUG_)
	cout << "GPU_DEVICE_ID               " <<  gpu_device_id << "="<< *gpu_device_id << endl;
	cout << "HETEROSCEDACTIC_MODE        " <<  heteroscedastic_mode <<"="<< *heteroscedastic_mode << endl;
	cout << "NQTL                        " <<  nqtlPtr << "="<<*nqtlPtr << endl;
	cout << "SISQUARE                    " <<  sigsquare << "="<<*sigsquare << endl;
	cout << "nLevelFixPtr                " <<  nLevelFixPtr << "="<<*nLevelFixPtr << endl;
	cout << "nLevelnLevelVarPtrFixPtr    " <<  nLevelVarPtr << "="<<*nLevelVarPtr << endl;
	cout << "corrLevelColPtr             " <<  corrLevelColPtr << "="<<*corrLevelColPtr << endl;
	cout << "work_cuda                   " <<  work_cuda << endl;
	cout << "bestim                      " <<  bestim << "="<<*bestim << endl;
	cout << "lrt                         " <<  lrt << "="<<*lrt << endl;
	cout << "osigsquare                  " <<  osigsquare << "="<<*osigsquare << endl;
	cout << "nsimPtr                     " <<  nsimPtr << "="<<*nsimPtr << endl;
#endif
	int  mode = *heteroscedastic_mode;

	QTLMapStructDeviceDataLinear      data ;
	QTLMapStructDeviceWorkLinear      work ;
	QTLMapStructDeviceSolutionLinear  solution ;

	cout << "****************** INFO DEVICES ******************************" << endl ;
	int nbDevice ;
	hipGetDeviceCount(&nbDevice);
	cout << "number of device    : " << nbDevice << endl ;

	hipDeviceProp_t  prop ;

	for (int i=0;i<nbDevice;i++) {
		hipGetDeviceProperties(&prop,i);
		cout << "id:"<< i << " ******* ===  "<< prop.name << "  ===  *******"<< endl ;
	}
	cout << "***************************************************************" << endl ;

	if ( *gpu_device_id >= nbDevice || *gpu_device_id < 0 ) {
		cerr << " bad gpu_divice_id : " << *gpu_device_id << endl ;
		exit(1);
	}

	/* Choose the device */
	hipGetDeviceProperties(&work.prop,*gpu_device_id);
	safecall(hipSetDevice(*gpu_device_id));


	cout << "Device Num:"<<  gpu_device_id << " - "<< work.prop.name << endl ;
	cout << "  totalGlobalMem : "      << prop.totalGlobalMem     << endl ;
	cout << "  sharedMemPerBlock : "   << prop.sharedMemPerBlock  << endl ;
	cout << "  regsPerBlock : "        << prop.regsPerBlock       << endl ; 
	cout << "  warpSize : "            << prop.warpSize           << endl ;
	cout << "  maxThreadsPerBlock : "  << prop.maxThreadsPerBlock << endl;
	cout << "  maxThreadsDim : "       << prop.maxThreadsDim[0]<< "," << prop.maxThreadsDim[1] << "," << prop.maxThreadsDim[2] << endl ;
	cout << "  maxGridSize : "         << prop.maxGridSize[0] << "," << prop.maxGridSize[1] << "," << prop.maxGridSize[2] << endl ;
	cout << "  clockRate : " << prop.clockRate << endl ;
	cout << "  totalConstMem : " << prop.totalConstMem << endl ;
	cout << "  multiProcessorCount : " << prop.multiProcessorCount << endl ;

	safecall(hipMemcpyToSymbol(HIP_SYMBOL(constNbLevelFix), nLevelFixPtr, sizeof(int)));
	safecall(hipMemcpyToSymbol(HIP_SYMBOL(constNbLevelVar), nLevelVarPtr, sizeof(int)));
	safecall(hipMemcpyToSymbol(HIP_SYMBOL(constNSim), nsimPtr, sizeof(int)));
	/*
	if ( ( (*nLevelFixPtr) + (*nLevelVarPtr)) >= MAXNTNIV_LOCALTHREAD_MEMORY ) {
		cerr << "Can not initialized cuda structure => update constant MAXNTNIV_LOCALTHREAD_MEMORY :"<<  MAXNTNIV_LOCALTHREAD_MEMORY << "<=" <<  (*nLevelFixPtr + *nLevelVarPtr) << endl ;
		exit(1);
	}	
	 */
	//	safecall(hipMemcpyToSymbol(HIP_SYMBOL(constCorrIndexCol), corrLevelColPtr, (*nLevelFixPtr + *nLevelVarPtr)*sizeof(DT)));
	size_t size=(*nLevelFixPtr + *nLevelVarPtr)*sizeof(DT);
	safecall(hipMalloc(&data.constCorrIndexCol,size));
	size_t size2=(*nLevelFixPtr + *nLevelVarPtr)*sizeof(DT);
	safecall(hipMemcpy(data.constCorrIndexCol, corrLevelColPtr, size2,hipMemcpyHostToDevice));

	if ( (*npPtr) >=  MAXNP_WORK ) {
		cerr << "Can not initialized cuda structure => update constant MAXNP_WORK :"<<  MAXNP_WORK << "<=" <<  (*npPtr) << endl ;
		exit(1);
	}

	safecall(hipMemcpyToSymbol(HIP_SYMBOL(constSizeFam),sizeFamilyNp,(*npPtr)*sizeof(int)));



	int err ;

	DT *xinc_d_var,*xinc_d_fix;

	/* Initialisation des donnees fixes a la position de la matrice de contingence */
	safecall(hipHostMalloc(& xinc_d_fix, (*ndPtr)*((*nLevelFixPtr)*sizeof(DT))));
	get_partialXincFix(work_cuda,xinc_d_fix);

	data.init(heteroscedastic_mode,nqtlPtr,
			sigsquare,xinc_d_fix,Y_d,
			CD_d,corrLevelColPtr,seuil_choPtr,
			ndPtr,nkdPtr,nsimPtr,npositionPtr,
			nLevelFixPtr,nLevelVarPtr,npPtr,sizeFamilyNp);




	int nbPosBlock;
	nbPosBlock = data.calculBlockPositionWorkSize(mode);
	if ( nbPosBlock > *npositionPtr ) {
		nbPosBlock = *npositionPtr ;
	} 

	data.npos = nbPosBlock ;
	work.initResolution(mode,data,CD_d,sigsquare,sizeFamilyNp) ;
	solution.init(data);

	/*
	 * 
	 *    Boucle des traitements par block
	 *    Utilisation de stream cuda
	 * 
	 * 
	 */
	/* Variables de travail pour recolter les resultats intermediaires */
	DT *local_osigsquare;
	DT *local_bestim;//[nbPosBlock*data.nsim*data.ntniv];

	safecall(hipHostMalloc(& local_osigsquare, nbPosBlock*data.nsim*data.np*sizeof(DT)));
	safecall(hipHostMalloc(& local_bestim, nbPosBlock*data.nsim*data.ntniv*sizeof(DT)));

	/* Creation des streams */
	int nstream=2;
	hipStream_t stream[nstream];
	for (int i=0;i<nstream;i++) {
		hipStreamCreate(&stream[i]);
	}
	int istream = 0 ;
	/* premier appel */

	int sizeBlockXincDVar = 0; /* taille en nombre de position a teste sur le groupe de liaison */
	int nbBlockXincDVar=2; /* On gere 2 block .l'un a remplir, l autre pour le calcul */
	int currentBlockMod = 0; 

	if ((*nLevelVarPtr) > 0) {
		/* Initialisation des donnees variables a la position de la matrice de contingence */
		sizeBlockXincDVar = (*ndPtr)*((*nLevelVarPtr)*nbPosBlock) ;
		safecall(hipHostMalloc(& xinc_d_var, nbBlockXincDVar*sizeBlockXincDVar*sizeof(DT)));
	}

	int lastPosition = 1;
	int nextPosition = nbPosBlock;
	get_partialXincVar(work_cuda,&lastPosition,&nextPosition,xinc_d_var);
	print_info_memory();

	QTLMapGenericModelCalcul *model ;

	switch (mode) {
	case QTLMapStructDeviceDataLinear::MODEL_HETERO_POLYGENIC :
		model = new QTLMapHeteroscedasticModelCalcul(data.npos,data.nsim,data.np);
		break;
	case QTLMapStructDeviceDataLinear::MODEL_HOMO_POLYGENIC :
		model = new QTLMapHomoscedasticModelCalcul();
		break;
	case QTLMapStructDeviceDataLinear::MODEL_HOMO_ANIMAL :
		model = new QTLMapHomoscedasticAnimalModelCalcul(data.nd,data.nLevelFix,M_d);
		break;
	default :
		cerr << "Devel error . not implementation of mode :"<< mode << endl ;
		exit(1);
	}

	/*
	 * Initialisation pour benchmark
	 */	
#if(_CUDA_HOST_TIME_PROF_)
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif

	for (int lastPosition = 1;lastPosition<=data.nposGlobal;lastPosition += nbPosBlock){   
#pragma omp sections
		{
#pragma omp section
			{
				int lastPositionNext = lastPosition + nbPosBlock ;
				if ( lastPositionNext <=data.nposGlobal ) {
					int nextPosition = lastPositionNext + nbPosBlock - 1;
					if ( nextPosition > data.nposGlobal) {
						nextPosition = data.nposGlobal;
					}

					int nextBlockMod = (currentBlockMod+1)%nbBlockXincDVar;
					get_partialXincVar(work_cuda,&lastPositionNext,&nextPosition,xinc_d_var+(nextBlockMod)*sizeBlockXincDVar);
				}
			}
#pragma omp section
			{
				nextPosition = lastPosition + nbPosBlock - 1;
				if ( nextPosition > data.nposGlobal) {
					nextPosition = data.nposGlobal;
				}
				data.npos = nextPosition - lastPosition + 1 ;

				cout << "   ** computing from position:"<< lastPosition << " at:"<< nextPosition << " (NPOS:"<< data.nposGlobal << ")   **" << endl << endl;

				/* Type de block utilise par les kernels */
				int nbblock_512 = ceil(data.npos / MAX_BLOCKDIM_512) + 1 ;
				int nbblock_64  = ceil(data.npos / MAX_BLOCKDIM_64) + 1 ;

				data.set_contingence_var(*ndPtr,*nLevelVarPtr,data.npos,0,xinc_d_var+currentBlockMod*sizeBlockXincDVar,&stream[istream]);//

				err = (int)hipInit(0);
				if ( err ) {
					printf(" hipInit error : %d \n",err);
					exit(1);
				}

				do {   /*  START IF CONVERGENCE IS OK */
					/* initializing model */
					model->start_analysis(stream[istream],data,work);
					hipStreamSynchronize(stream[istream]);
#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(start, 0);
#endif		
					model->calcul_XT_X_A(stream[istream],data,work);
					/* on peut peut etre enlever cette synchronisation et la mettre plus tard */
					//hipStreamSynchronize(stream[istream]);
#if(_CUDA_HOST_TIME_PROF_)
					hipStreamSynchronize(stream[istream]);
					hipEventRecord(stop, 0);
					hipEventElapsedTime(&elapsedTime, start, stop);
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
					cout << "[BENCHMARK]  XT_X_A grid(x,y,z): elapsedtime:"<< elapsedTime<< endl ;
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
#endif
#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(start, 0);
#endif				
					model->calcul_XT_X_B(stream[istream],data,work);
					/* on peut peut etre enlever cette synchronisation et la mettre plus tard */
					//hipStreamSynchronize(stream[istream]);
					safecall(hipGetLastError());

#if(_CUDA_HOST_TIME_PROF_)
					hipStreamSynchronize(stream[istream]);
					hipEventRecord(stop, 0);
					hipEventElapsedTime(&elapsedTime, start, stop);
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
					cout << "[BENCHMARK]  XT_X_B grid(x,y,z): elapsedtime:"<< elapsedTime<< endl ;
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
#endif


#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(start, 0);
#endif
					model->calcul_XT_X_C(stream[istream],data,work);
					hipStreamSynchronize(stream[istream]);
					safecall(hipGetLastError());


#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(stop, 0);
					hipEventElapsedTime(&elapsedTime, start, stop);
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
					cout << "[BENCHMARK]  XT_X_C grid(x,y,z) elapsedtime:"<< elapsedTime<< endl ;
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
#endif

#if(_CUDA_HOST_DEBUG_)
					printMatSymXXTriang(model->getType(),data,work.XX);
#endif

#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(start, 0);
#endif
					/* Cholesky decomp */
					model->calcul_Cholesky_Decomposition(stream[istream],data,work);
					hipStreamSynchronize(stream[istream]);
					safecall(hipGetLastError());

					//	printMatSymXXTriang(model->getType(),data,work.triang);

#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(stop, 0);
					hipEventElapsedTime(&elapsedTime, start, stop);
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
					cout << "[BENCHMARK]  Cholesky_Decomposition grid(x,y,z) elapsedtime:"<< elapsedTime<< endl ;
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
#endif

#if(_CUDA_HOST_DEBUG_)
					printf("Cholesky decomp\n----------------------------------------------------------\n");
					printMatSymXXTriang(model->getType(),data,work.triang);
#endif

#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(start, 0);
#endif								
					model->calcul_RHS(stream[istream],data,work);
					hipStreamSynchronize(stream[istream]);
					safecall(hipGetLastError());

#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(stop, 0);
					hipEventElapsedTime(&elapsedTime, start, stop);
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
					cout << "[BENCHMARK]  Set_RHS_FIX + Set_RHS_VAR grid(x,y,z):"<< " elapsedtime:"<< elapsedTime<< endl ;
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
#endif


#if(_CUDA_HOST_DEBUG_)
					printf("RHS\n-------POS=1,SIM=nsim => NTNIV---------------------------------------------------\n");
					Utils<DT>::printFloatDeviceArray3D(data.npos,data.nsim,data.ntniv,10,1,data.ntniv,work.rhs);
#endif		

#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(start, 0);
#endif
					/* Solution */
					model->calcul_LU(stream[istream],data,work,solution);
					hipStreamSynchronize(stream[istream]);
					safecall(hipGetLastError());

#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(stop, 0);
					hipEventElapsedTime(&elapsedTime, start, stop);
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
					cout << "[BENCHMARK]  Resolve_LU grid(x,y,z): elapsedtime:"<< elapsedTime<< endl ;
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
#endif


#if(_CUDA_HOST_DEBUG_)
					printf("Resolution LU\n------POS=1,SIM=nsim => NTNIV ----------------------------------------------------\n");
					Utils<DT>::printFloatDeviceArray3D(data.npos,data.nsim,data.ntniv,4,1,data.ntniv,solution.bestim);
#endif

					/* Recuperation des resultats */
					Utils<DT>::getArrayDeviceToHost(nbblock_512,MAX_BLOCKDIM_512,data.npos*data.nsim*data.ntniv,solution.bestim,local_bestim,&stream[istream]);

#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(start, 0);
#endif
					/* Variances residuelles */
					model->calcul_SIGSQ(stream[istream],data,work,solution);
					/* on synchronise pour que local_bestim soit initialise */
					hipStreamSynchronize(stream[istream]);

#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(stop, 0);
					hipEventElapsedTime(&elapsedTime, start, stop);
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
					cout << "[BENCHMARK]  Set_SIGSQ grid(x,y,z)  elapsedtime:"<< elapsedTime<< endl ;
					cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
#endif

					for (int ipos=0;ipos<data.npos;ipos++) {
						for (int isim=0;isim<data.nsim;isim++) {
							for (int iniv=0;iniv<data.ntniv;iniv++) {
								bestim[ipos+lastPosition-1+isim*data.nposGlobal+iniv*data.nsim*data.nposGlobal] = local_bestim[ipos+isim*data.npos+iniv*data.nsim*data.npos] ;
							}
						}
					}

					hipStreamSynchronize(stream[istream]);
					safecall(hipGetLastError());

#if(_CUDA_HOST_DEBUG_)
					cout << "SIGSQ------------POS=1,SIM=nsim => NP----------------------------------------------"<<endl ;
					Utils<DT>::printFloatDeviceArray3D(data.npos,data.nsim,data.np,data.npos,1,data.np,solution.osigsq);
#endif
					/* Recuperation des resultats */
					Utils<DT>::getArrayDeviceToHost(nbblock_512,MAX_BLOCKDIM_512,data.npos*data.nsim*data.np,solution.osigsq,local_osigsquare,&stream[istream]);



#if(_CUDA_HOST_TIME_PROF_)
					hipEventRecord(start, 0);
#endif
					hipStreamSynchronize(stream[istream]);
				} while ( ! model->convergenceOk(stream[istream],data,work,solution) ); /* FIN CONVERGENCE */

				if (data.nqtl > 0 ) {
					dim3 dimBlockLrt(32,16);
					dim3 dimGridLrt (ceil(data.npos/dimBlockLrt.x)+1 , data.np );
					/* Calcul LRT */
					Set_LRT<<<dimGridLrt,dimBlockLrt,0,stream[istream]>>>(data,solution.osigsq,solution.lrt,lastPosition-1);

#if(_CUDA_HOST_DEBUG_)
					printf("PARTIEL LRT\n------------POS=1,SIM=1,..,nsim => NP----------------------------------------------\n");
					hipStreamSynchronize(stream[istream]);
					Utils<DT>::printFloatDeviceArray3D(data.nposGlobal,data.nsim,data.nqtl,data.nposGlobal,1,data.nqtl,solution.lrt);
#endif
					safecall(hipGetLastError());
					hipStreamSynchronize(stream[istream]);
				}


#if(_CUDA_HOST_TIME_PROF_)
				hipEventRecord(stop, 0);
				hipEventElapsedTime(&elapsedTime, start, stop);
				cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
				cout << "[BENCHMARK]  Set_LRT grid(x,y,z)  elapsedtime:"<< elapsedTime<< endl ;
				cout << "----------------------------------------------------------------------------------------------------------------"<< endl ;
#endif



				for (int ipos=0;ipos<data.npos;ipos++) {
					for (int isim=0;isim<data.nsim;isim++) {
						for (int ip=0;ip<data.np;ip++) {
							osigsquare[ipos+lastPosition-1+isim*data.nposGlobal+ip*data.nsim*data.nposGlobal] = local_osigsquare[ipos+isim*data.npos+ip*data.nsim*data.npos];//[ip+isim*data.np+ipos*data.nsim*data.np] ;
						}
					}
				}
			}
		}
		istream = (istream+1)%nstream;
		currentBlockMod = (currentBlockMod+1)%nbBlockXincDVar ;

	}

	/* Fin boucle principale  */

	for (int i=0;i<nstream;i++) {
		hipStreamDestroy(stream[i]);
	}


	work.releaseDeviceResolution();
	safecall(hipHostFree(xinc_d_fix));
	if ( data.nqtl>=1 ) safecall(hipHostFree(xinc_d_var));
	safecall(hipHostFree(local_osigsquare));
	safecall(hipHostFree(local_bestim));

	if ( data.nqtl>=1 ) {	
		/* Recuperation des LRTs */
		int nbblock_512_global = ceil(data.nposGlobal / MAX_BLOCKDIM_512) + 1 ;
		Utils<DT>::getArrayDeviceToHost(nbblock_512_global,MAX_BLOCKDIM_512,data.nqtl*data.nposGlobal*data.nsim*data.np,solution.lrt,lrt,NULL);

//		for (int ip=0;ip<data.np;ip++) {
//			cout << "IP="<< ip << endl ;
//			for (int i=0;i<data.nposGlobal;i++) {
//				cout << lrt[i+ip*data.nqtl*data.nposGlobal*data.nsim]<< " ";
//			}
//			cout << endl ;
//		}

		for (int isim=0;isim<data.nsim;isim++) {
			for (int iq=0;iq<data.nqtl;iq++) {
				maxLRT[isim+iq*data.nsim]      = -999.9;
			}
			maxPosition[isim] = -1;


			for (int i=0;i<data.nposGlobal;i++) {
				DT lrtmax = 0 ;
				for (int ip=0;ip<data.np;ip++) {
					lrtmax += lrt[i+isim*data.nposGlobal+(data.nqtl-1)*data.nposGlobal*data.nsim+ip*data.nqtl*data.nposGlobal*data.nsim];
				}
				if ( maxLRT[isim+(data.nqtl-1)*data.nsim] < lrtmax ) {
					for (int iq=0;iq<data.nqtl;iq++) {
						DT lrtmax = 0 ;
						for (int ip=0;ip<data.np;ip++) {
							lrtmax += lrt[i+isim*data.nposGlobal+iq*data.nposGlobal*data.nsim+ip*data.nqtl*data.nposGlobal*data.nsim];
						}
						maxLRT[isim+iq*data.nsim] = lrtmax;
					}
					maxPosition[isim] = i+1 ; /* index pour fortran */
				}
			} // fin i
	} // fin isim

	add_internal_sig(osigsquare,maxPosition,data.nposGlobal,data.nsim,data.np,data.nqtl);
} else { /* residual variance under H0 */
	int *maxP;
	safecall(hipHostMalloc(&maxP,data.nsim*sizeof(DT)));
	for (int isim=0;isim<data.nsim;isim++)maxP[isim]=1;
	add_internal_sig(osigsquare,maxP,data.nposGlobal,data.nsim,data.np,data.nqtl);
	safecall(hipHostFree(maxP));
}
delete model;

#if(_CUDA_HOST_TIME_PROF_)
hipEventDestroy(start);
hipEventDestroy(stop);
#endif

#if(_CUDA_HOST_DEBUG_)
cout << "DATA RELEASED ON DEVICE" << endl ;
#endif
data.releaseDevice();

#if(_CUDA_HOST_DEBUG_)
cout << "SOLUTION RELEASED ON DEVICE" << endl ;
#endif
solution.releaseDevice();

#if(_CUDA_HOST_DEBUG_)
cout << " ** END ** " << endl ;
#endif

/* Modif OFI Mai 2012 constCorrIndexCol n est plus dans la memoire constante (ne fonctionnait pas quand le nombre de niveau atteignait 160)
 * cet appel devra etre mis dans data.releasedevice() */
safecall(hipFree(data.constCorrIndexCol));

}
